#include "hip/hip_runtime.h"
// Ttt
#include <cstdlib>

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include <math.h>

#include "../src/graphcode.h"
#include "../src/cudahelper.cuh"
#include "../src/helper.h"


#define BLOCKSIZE_x 16
#define BLOCKSIZE_y 16

#define Nrows 4
#define Ncols 4

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int hostPtr, int b){ return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b); }



__global__ void check(int *data, int *comparedata, unsigned long matrixSize, int *pInt, int *pInt1) {
    int tid = blockIdx.x;

     //int q = sqrt((float)matrixSize);

     for (int i = 0; i < matrixSize; i++) {
        if (tid == i*matrixSize+i) {
            //Can be used to debug
            //pInt[tid] = -1;
            return;
        }
     }

    if (data[tid] != 0 ) {
        if (comparedata[tid] != 0) {
            pInt1[tid] = 1;
            if (data[tid] == comparedata[tid]) {
                pInt[tid] = 1;
            }

        }
    }

}



/******************/
/* TEST KERNEL 2D */
/******************/
__global__ void test_kernel_2D(float *devPtr, size_t pitch)
{
    int    tidx = blockIdx.x*blockDim.x + threadIdx.x;
    int    tidy = blockIdx.y*blockDim.y + threadIdx.y;

    if ((tidx < Ncols) && (tidy < Nrows))
    {
        float *row_a = (float *)((char*)devPtr + tidy * pitch);
        if (tidx == tidy) {
            row_a[tidx] = 0.0;
        } else {

            row_a[tidx] = row_a[tidx] * tidx * tidy;
        }
    }
}

/********/
/* MAIN */
/********/
int testCudaMatrixMemory()
{
    float hostPtr[Nrows][Ncols];
    float *devPtr;
    size_t pitch;

    for (int i = 0; i < Nrows; i++)
        for (int j = 0; j < Ncols; j++) {
            hostPtr[i][j] = 1.f;
            //printf("row %i column %i value %f \n", i, j, hostPtr[i][j]);
        }

    // --- 2D pitched allocation and host->device memcopy
    HANDLE_ERROR(hipMallocPitch(&devPtr, &pitch, Ncols * sizeof(float), Nrows));
    HANDLE_ERROR(hipMemcpy2D(devPtr, pitch, hostPtr, Ncols*sizeof(float), Ncols*sizeof(float), Nrows, hipMemcpyHostToDevice));

    dim3 gridSize(iDivUp(Ncols, BLOCKSIZE_x), iDivUp(Nrows, BLOCKSIZE_y));
    dim3 blockSize(BLOCKSIZE_y, BLOCKSIZE_x);

    test_kernel_2D<<<gridSize, blockSize>>>(devPtr, pitch);

    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipMemcpy2D(hostPtr, Ncols * sizeof(float), devPtr, pitch, Ncols * sizeof(float), Nrows, hipMemcpyDeviceToHost));

    for (int i = 0; i < Nrows; i++)
        for (int j = 0; j < Ncols; j++)
            printf("row %i column %i value %f \n", i, j, hostPtr[i][j]);

    return 0;
}

void testCudaLinearMatrixMemory(){
    nlohmann::json gcq;
    gcq["dictionary"] = { "head", "body"};
    gcq["matrix"] = {{1,1}, {0,1}};

    json gc1Dictionary = gcq["dictionary"];

    int matrix1[gc1Dictionary.size()][gc1Dictionary.size()];
    convertDict2Matrix(gc1Dictionary.size(), (int *) matrix1, gcq["matrix"]);

    int inputMatrix[gcq.size() * gcq.size()];
    int count = 0;
    for (int i = 0; i < gcq.size(); i++)
        for (int j = 0; j < gcq.size(); j++) {
            inputMatrix[count++] = matrix1[i][j];
        }

    int *gpu_inputMatrix;
    int *darr_edge_metric_count;
    int *darr_num_of_non_zero_edges;
    // Allocate device memory for inputMatrix
    //hipMalloc((void**)&gpu_inputMatrix, sizeof(int) );

    int items = 4;


    HANDLE_ERROR(hipMalloc((void**)&gpu_inputMatrix, sizeof(int) * items) );
    HANDLE_ERROR(hipMalloc((void**)&darr_edge_metric_count, sizeof(int) * items) );
    HANDLE_ERROR(hipMalloc((void**)&darr_num_of_non_zero_edges, sizeof(int) * items) );
    /*
    hipMemcpy2DToArray (dst,
                         0,
                         0,
                         matrix1,
                         sizeof(int),
                         gc1Dictionary.size() * sizeof(int),
                         gc1Dictionary.size(),
                         hipMemcpyHostToDevice );

    */

    // Transfer data from host to device memory
    HANDLE_ERROR(hipMemcpy(gpu_inputMatrix, inputMatrix, sizeof(int) * gcq.size() * gcq.size(), hipMemcpyHostToDevice));


    check<<<items, items>>>(gpu_inputMatrix, gpu_inputMatrix, gcq.size(), darr_edge_metric_count,
                            darr_num_of_non_zero_edges);


    int arr_edge_metric_count[items];
    int arr_num_of_non_zero_edges[items];

    HANDLE_ERROR(hipMemcpy(arr_edge_metric_count, darr_edge_metric_count, sizeof (int) * gcq.size() * gcq.size(), hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(arr_num_of_non_zero_edges, darr_num_of_non_zero_edges, sizeof (int) * gcq.size() * gcq.size(), hipMemcpyDeviceToHost));

    int num_of_non_zero_edges = 0;
    int edge_metric_count = 0;
    for(int i = 0; i < items; i++) {
        std::cout << "pos: " << i << " value: " << arr_edge_metric_count[i] << std::endl;
        if (arr_edge_metric_count[i] == 1) {
            edge_metric_count++;
        }
        if (arr_num_of_non_zero_edges[i] == 1) {
            num_of_non_zero_edges++;
        }
    }
    float node_metric = (float) items/2 / (float) gc1Dictionary.size();

    float edge_metric = 0.0;
    if (num_of_non_zero_edges > 0)
        edge_metric = (float) edge_metric_count / (float) num_of_non_zero_edges;


    std::cout << "Similarity: " << " value: " << node_metric << std::endl;
    std::cout << "Recommendation: " << " value: " << edge_metric << std::endl;



    HANDLE_ERROR(hipFree(gpu_inputMatrix));
}


int main(int, char**)
{
    int q = sqrt((float)4);

    for (int i = 0; i < q; i++) {

            std::cout <<i*q+i << std::endl;
    }

    testCudaMatrixMemory();
    testCudaLinearMatrixMemory();

}
