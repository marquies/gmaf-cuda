#include "hip/hip_runtime.h"
//
// Created by breucking on 06.11.21.
//



#include <cstdlib>


#include "../src/graphcode.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<>

#define BLOCKSIZE_x 16
#define BLOCKSIZE_y 16

#define Nrows 4
#define Ncols 4

/*****************/
/* CUDA MEMCHECK */
/*****************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
        if (abort) { exit(code); }
    }
}

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int hostPtr, int b){ return ((hostPtr % b) != 0) ? (hostPtr / b + 1) : (hostPtr / b); }

/******************/
/* TEST KERNEL 2D */
/******************/
__global__ void test_kernel_2D(float *devPtr, size_t pitch)
{
    int    tidx = blockIdx.x*blockDim.x + threadIdx.x;
    int    tidy = blockIdx.y*blockDim.y + threadIdx.y;



    if ((tidx < Ncols) && (tidy < Nrows))
    {
        float *row_a = (float *)((char*)devPtr + tidy * pitch);
        if (tidx == tidy) {
            row_a[tidx] = 0.0;
        } else {

            row_a[tidx] = row_a[tidx] * tidx * tidy;
        }
    }
}

/********/
/* MAIN */
/********/
int testcuda()
{
    float hostPtr[Nrows][Ncols];
    float *devPtr;
    size_t pitch;

    for (int i = 0; i < Nrows; i++)
        for (int j = 0; j < Ncols; j++) {
            hostPtr[i][j] = 1.f;
            //printf("row %i column %i value %f \n", i, j, hostPtr[i][j]);
        }

    // --- 2D pitched allocation and host->device memcopy
    gpuErrchk(hipMallocPitch(&devPtr, &pitch, Ncols * sizeof(float), Nrows));
    gpuErrchk(hipMemcpy2D(devPtr, pitch, hostPtr, Ncols*sizeof(float), Ncols*sizeof(float), Nrows, hipMemcpyHostToDevice));

    dim3 gridSize(iDivUp(Ncols, BLOCKSIZE_x), iDivUp(Nrows, BLOCKSIZE_y));
    dim3 blockSize(BLOCKSIZE_y, BLOCKSIZE_x);

    test_kernel_2D<<<gridSize, blockSize>>>(devPtr, pitch);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy2D(hostPtr, Ncols * sizeof(float), devPtr, pitch, Ncols * sizeof(float), Nrows, hipMemcpyDeviceToHost));

    for (int i = 0; i < Nrows; i++)
        for (int j = 0; j < Ncols; j++)
            printf("row %i column %i value %f \n", i, j, hostPtr[i][j]);

    return 0;
}
void testBasic()
{
    // do some nice calculation; store the results in `foo` and `bar`,
    // respectively

    nlohmann::json gcq;
    gcq["dictionary"] = { "head", "body"};
    gcq["matrix"] = {{1,1}, {0,1}};


    std::vector<json> others;
    others.push_back(gcq);

    gmaf::GraphCode gc;

    gc.calculateSimilarityV(0,&gcq,&others,0,1);

    //exit(17);
    //ALEPH_ASSERT_THROW( foo != bar );
    //ALEPH_ASSERT_EQUAL( foo, 2.0 );
    //ALEPH_ASSERT_EQUAL( bar, 1.0 );
}

void testAdvanced()
{
    // a more advanced test
}

int main(int, char**)
{
    testBasic();
    testAdvanced();
    testcuda();
}
