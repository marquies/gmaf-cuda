#include "hip/hip_runtime.h"
//#include <graphcode.h>
//#include <cudahelper.cuh>
#include <hip/hip_runtime.h>

#include <c++/9/iostream>
#include "../src/cuda_algorithms.cuh"
#include "../src/cudahelper.cuh"


////
//// Created by breucking on 31.01.22.
////
//
//
//
//
//
////defines the shared memory size
//#include <cudahelper.cuh>
//#include <cassert>
//#include <c++/9/iostream>
//#include <graphcode.h>
//
//#define SHARED_LIMIT 1024
//
//#define GIGA 1073741824
///*
// * division of the vector to be sorted in buckets
// * the attributes of the object Block are the parameters of each bucket
// */
//template <typename Type>
//struct Block
//{
//
//    unsigned int begin;
//    unsigned int end;
//
//    unsigned int nextbegin;
//    unsigned int nextend;
//
//    Type		 pivot;
//
//    //max of the bucket items
//    Type		 maxPiv;
//    //min of the bucket items
//    Type		 minPiv;
//    //done indicates that a bucket has been analyzed
//    short		 done;
//    short		 select;
//
//
//};
//
//
//
//template <typename Type>
//struct Partition
//{
//
//    unsigned int ibucket;
//    unsigned int from;
//    unsigned int end;
//    Type pivot;
//};
//
//typedef unsigned int uint;
//
//size_t scanInclusiveShort(
//        uint *d_Dst,
//        uint *d_Src,
//        uint batchSize,
//        uint arrayLength
//);
//
//size_t scanInclusiveLarge(
//        uint *d_Dst,
//        uint *d_Src,
//        uint batchSize,
//        uint arrayLength
//);
//
//
//void CUDA_Quicksort(Metrics *inData, Metrics *outData, unsigned int dataSize, unsigned int threads /*, int Device, double* timer*/);
//
//void CUDA_Quicksort_64(double* inData,double* outData, unsigned int size, unsigned int threads, int Device, double* timer);
//
//typedef unsigned int Type;
//
//void test_bitonicSort(unsigned int* h_InputKey,unsigned int N, double* timer);
//void test_MergeSort  (unsigned int*h_SrcKey   ,unsigned int N, double* timer);
//void test_thrustSort (Type* h_data    ,unsigned int N, double* timer);
//
//
void testCudaSort();
//
//int main() {
//    testCudaSort();
//}

////////////////////////////////////////////////////////////////////////////////
// Initialize data on the host.
////////////////////////////////////////////////////////////////////////////////
void initialize_data(unsigned int *dst, unsigned int nitems) {
    // Fixed seed for illustration
    srand(2047);

    // Fill dst with random values
    for (unsigned i = 0; i < nitems; i++)
        dst[i] = rand() % nitems;
}
////////////////////////////////////////////////////////////////////////////////
// Main entry point.
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    int num_items = 128;
    bool verbose = false;


    verbose = true;
    num_items = 100000;



    // Get device properties
    int device_count = 0, device = 0;
    hipDeviceProp_t properties;
    HANDLE_ERROR(hipGetDeviceProperties(&properties, device));

    if (properties.major > 3 || (properties.major == 3 && properties.minor >= 5)) {
        std::cout << "Running on GPU " << device << " (" << properties.name << ")" << std::endl;
    } else {
        std::cout << "ERROR: cdpsimpleQuicksort requires GPU devices with compute SM 3.5 or higher." << std::endl;
        std::cout << "Current GPU device has compute SM" << properties.major << "." << properties.minor
                  << ". Exiting..." << std::endl;
        exit(EXIT_FAILURE);
    }


    if (device == -1) {
        std::cerr << "cdpSimpleQuicksort requires GPU devices with compute SM 3.5 or higher.  Exiting..." << std::endl;
        exit(EXIT_SUCCESS);
    }

    hipSetDevice(device);

    // Create input data
    unsigned int *h_data = 0;
    unsigned int *d_data = 0;

    // Allocate CPU memory and initialize data.
    std::cout << "Initializing data:" << std::endl;
    h_data = (unsigned int *) malloc(num_items * sizeof(unsigned int));
    initialize_data(h_data, num_items);

//    if (verbose) {
//        for (int i = 0; i < num_items; i++)
//            std::cout << "Data [" << i << "]: " << h_data[i] << std::endl;
//    }

    // Allocate GPU memory.
    HANDLE_ERROR(hipMalloc((void **) &d_data, num_items * sizeof(unsigned int)));
    HANDLE_ERROR(hipMemcpy(d_data, h_data, num_items * sizeof(unsigned int), hipMemcpyHostToDevice));

    // Execute
    std::cout << "Running quicksort on " << num_items << " elements" << std::endl;
    run_qsort(d_data, num_items);

    // Check result
    std::cout << "Validating results: ";
    check_results(num_items, d_data);

    free(h_data);
    HANDLE_ERROR(hipFree(d_data));

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();
    exit(EXIT_SUCCESS);
}

//
void testCudaSort() {
//
//    const unsigned int N = 100000;
//    Metrics *inData = new Metrics[N];
//
//    for (int i = 0; i < N; i++) {
//        inData[i].idx = N - i;
//    }
//
//    for (int i = 0; i < N; i++) {
//        // std::cout << "(" << i <<") " << inData[i] << " " ;
//    }
//    Metrics *outData = new Metrics[N];
//    unsigned int num_items = N;
////    CUDA_Quicksort(inData,outData, N,128);
//
//
//    for (int i = 0; i < N; i++) {
//        std::cout << "(" << i << ") " << outData[i].idx << " ";
//    }


}
//
//extern __shared__ uint sMemory[];
//
//
//
////All three kernels run 512 threads per workgroup
////Must be a power of two
//#define THREADBLOCK_SIZE 256
//
//////////////////////////////////////////////////////////////////////////////////
//// Basic ccan codelets
//////////////////////////////////////////////////////////////////////////////////
//#if(0)
////Naive inclusive scan: O(N * log2(N)) operations
//    //Allocate 2 * 'size' local memory, initialize the first half
//    //with 'size' zeros avoiding if(pos >= offset) condition evaluation
//    //and saving instructions
//    inline __device__ uint scan1Inclusive(uint idata,  uint *s_Data, uint size){
//        uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
//        s_Data[pos] = 0;
//        pos += size;
//        s_Data[pos] = idata;
//
//        for(uint offset = 1; offset < size; offset <<= 1){
//            __syncthreads();
//            uint t = s_Data[pos] + s_Data[pos - offset];
//            __syncthreads();
//            s_Data[pos] = t;
//        }
//
//        return s_Data[pos];
//    }
//
//    inline __device__ uint scan1Exclusive(uint idata,  uint *s_Data, uint size){
//        return scan1Inclusive(idata, s_Data, size) - idata;
//    }
//
//#else
//#define LOG2_WARP_SIZE 5U
//#define WARP_SIZE (1U << LOG2_WARP_SIZE)
//
////Almost the same as naive scan1Inclusive, but doesn't need __syncthreads()
////assuming size <= WARP_SIZE
//inline __device__ uint warpScanInclusive(uint idata,  uint *s_Data, uint size){
//    uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
//    s_Data[pos] = 0;
//    pos += size;
//    s_Data[pos] = idata;
//
//    for(uint offset = 1; offset < size; offset <<= 1)
//        s_Data[pos] += s_Data[pos - offset];
//
//    return s_Data[pos];
//}
//
//inline __device__ uint warpScanExclusive(uint idata,  uint *s_Data, uint size){
//    return warpScanInclusive(idata, s_Data, size) - idata;
//}
//
//inline __device__ uint scan1Inclusive(uint idata,  uint *s_Data, uint size){
//    if(size > WARP_SIZE){
//        //Bottom-level inclusive warp scan
//        uint warpResult = warpScanInclusive(idata, s_Data, WARP_SIZE);
//
//        //Save top elements of each warp for exclusive warp scan
//        //sync to wait for warp scans to complete (because s_Data is being overwritten)
//        __syncthreads();
//        if( (threadIdx.x & (WARP_SIZE - 1)) == (WARP_SIZE - 1) )
//            s_Data[threadIdx.x >> LOG2_WARP_SIZE] = warpResult;
//
//        //wait for warp scans to complete
//        __syncthreads();
//        if( threadIdx.x < (THREADBLOCK_SIZE / WARP_SIZE) ){
//            //grab top warp elements
//            uint val = s_Data[threadIdx.x];
//            //calculate exclsive scan and write back to shared memory
//            s_Data[threadIdx.x] = warpScanExclusive(val, s_Data, size >> LOG2_WARP_SIZE);
//        }
//
//        //return updated warp scans with exclusive scan results
//        __syncthreads();
//        return warpResult + s_Data[threadIdx.x >> LOG2_WARP_SIZE];
//    }else{
//        return warpScanInclusive(idata, s_Data, size);
//    }
//}
//
//inline __device__ uint scan1Exclusive(uint idata,  uint *s_Data, uint size){
//    return scan1Inclusive(idata, s_Data, size) - idata;
//}
//
//#endif
//__device__ inline  double atomicMax(double* address, double val)
//{
//    unsigned long long int* address_as_ull = (unsigned long long int*)address;
//    unsigned long long int assumed;
//    unsigned long long int old = *address_as_ull;
//
//    assumed = old;
//    old = atomicCAS(address_as_ull,
//                    assumed,
//                    __double_as_longlong(max(val ,__longlong_as_double(assumed))));
//
//    while (assumed != old)
//    {
//        assumed = old;
//        old = atomicCAS(address_as_ull,
//                        assumed,
//                        __double_as_longlong(max(val ,__longlong_as_double(assumed))));
//    }
//    return __longlong_as_double(old);
//}
//
//
//__device__ inline double atomicMin(double* address, double val)
//{
//    unsigned long long int* address_as_ull = (unsigned long long int*)address;
//    unsigned long long int old = *address_as_ull, assumed;
//
//    assumed = old;
//    old = atomicCAS(address_as_ull,
//                    assumed,
//                    __double_as_longlong(min(val ,__longlong_as_double(assumed))));
//    while (assumed != old)
//    {
//        assumed = old;
//        old = atomicCAS(address_as_ull,
//                        assumed,
//                        __double_as_longlong(min(val ,__longlong_as_double(assumed))));
//    }
//    return __longlong_as_double(old);
//}
//
//
//
//
//
//template <typename Type>
//__device__ inline void Comparator(
//
//        Type& valA,
//        Type& valB,
//        uint dir
//){
//    Type t;
//
//    if( (valA > valB) == dir ){
//        t = valA; valA = valB; valB = t;
//    }
//}
////
////template <typename Type>
////__device__ inline void Comparator(
////// compare(Metrics *positionA, Metrics *positionB) {
////        Metrics positionA,
////        Metrics positionB,
////        uint dir
////){
////
////    //if ( array[ index ].similarity < array[ smallest ].similarity )
////    float a = positionA.similarity * 100000.0f + positionA.recommendation * 100.0f +
////              positionA.inferencing;
////
////
////    float b = positionB.similarity * 100000.0f + positionB.recommendation * 100.0f +
////              positionB.inferencing;
////
////
////        Metrics t;
////
////    if( (a-b > 0) == dir ){
////        t = positionA; positionA = positionB; positionB = t;
////    }
////
////    Type t;
////
////    if( (valA > valB) == dir ){
////        t = valA; valA = valB; valB = t;
////    }
////}
//
//
//
//
//static __device__ __forceinline__ unsigned int __qsflo(unsigned int word)
//{
//    unsigned int ret;
//    asm ("bfind.u32 %0, %1;" : "=r"(ret) : "r"(word));
//    return ret;
//}
//
//template <typename Type>
//__global__ void globalBitonicSort(Type* indata,Type*outdata, Block<Type>* bucket, bool inputSelect)
//{
//    __shared__ uint shared[1024];
//
//
//    Type* data;
//
//    Block<Type> cord = bucket[blockIdx.x];
//
//
//    uint size=cord.end-cord.begin;
//    bool select = !(cord.select);
//
//    if(cord.end-cord.begin>1024 || cord.end-cord.begin==0)
//        return;
//
//    unsigned int bitonicSize = 1 << (__qsflo(size-1U)+1);
//
//
//    if(select)
//        data = indata;
//    else
//        data = outdata;
//
//    //__syncthreads();
//
//    for(int i=threadIdx.x;i<size;i+=blockDim.x)
//        shared[i] = data[i+cord.begin];
//
//
//    for(int i=threadIdx.x+size;i<bitonicSize;i+=blockDim.x)
//        shared[i] = 0xffffffff;
//
//    __syncthreads();
//
//
//    for(uint size = 2; size < bitonicSize; size <<= 1){
//        //Bitonic merge
//        uint ddd = 1 ^ ( (threadIdx.x & (size / 2)) != 0 );
//        for(uint stride = size / 2; stride > 0; stride >>= 1){
//            __syncthreads();
//            uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
//            //if(pos <bitonicSize){
//            Comparator(
//                    shared[pos +      0],
//                    shared[pos + stride],
//                    ddd
//            );
//            // }
//        }
//    }
//
//
//    //ddd == dir for the last bitonic merge step
//
//    for(uint stride = bitonicSize / 2; stride > 0; stride >>= 1){
//        __syncthreads();
//        uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
//        // if(pos <bitonicSize){
//        Comparator(
//                shared[pos +      0],
//                shared[pos + stride],
//                1
//        );
//        // }
//    }
//
//    __syncthreads();
//
//    // Write back the sorted data to its correct position
//    for(int i=threadIdx.x;i<size;i+=blockDim.x)
//        indata[i+cord.begin] = shared[i];
//
//}
//
//
//
//template <typename Type>
//inline __device__ void warpScanInclusive2(Type& idata,Type& idata2,  Type *s_Data, Type *s_Data2, uint size){
//
//    // uint* s_Data2;
//    //s_Data2 = s_Data + blockDim.x*2;
//
//    uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
//    s_Data[pos] = 0;
//    s_Data2[pos] = 0;
//    pos += size;
//    s_Data[pos] = idata;
//    s_Data2[pos] = idata2;
//
//    for(uint offset = 1; offset < size; offset <<= 1)
//    {
//        s_Data[pos] += s_Data[pos - offset];
//        s_Data2[pos] += s_Data2[pos - offset];
//    }
//
//    idata=s_Data[pos];
//    idata2=s_Data2[pos];
//}
//
//template <typename Type>
//inline __device__ void warpScanExclusive2(Type& idata,Type& idata2,  Type *s_Data, Type *s_Data2, uint size){
//
//    // uint* s_Data2;
//    //s_Data2 = s_Data + blockDim.x*2;
//
//    uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
//    s_Data[pos] = 0;
//    s_Data2[pos] = 0;
//    pos += size;
//    s_Data[pos] = idata;
//    s_Data2[pos] = idata2;
//
//    for(uint offset = 1; offset < size; offset <<= 1)
//    {
//        s_Data[pos] += s_Data[pos - offset];
//        s_Data2[pos] += s_Data2[pos - offset];
//    }
//
//    idata=s_Data[pos]-idata;
//    idata2=s_Data2[pos]-idata2;
//}
//
//#define LOG2_WARP_SIZE 5U
//#define WARP_SIZE (1U << LOG2_WARP_SIZE)
//
//template <typename Type>
//inline __device__ void scan1Inclusive2(Type& idata,Type& idata2,  Type *s_Data, uint size){
//
//     Type* s_Data2;
//    s_Data2 = s_Data + blockDim.x*2;
//
//    if(size > WARP_SIZE){
//
//        //Bottom-level inclusive warp scan
//        warpScanInclusive2(idata,idata2, s_Data,s_Data2, WARP_SIZE);
//
//        //Save top Types of each warp for exclusive warp scan
//        //sync to wait for warp scans to complete (because s_Data is being overwritten)
//        __syncthreads();
//        if( (threadIdx.x & (WARP_SIZE - 1)) == (WARP_SIZE - 1) )
//        {
//            s_Data[threadIdx.x >> LOG2_WARP_SIZE] = idata;
//            s_Data2[threadIdx.x >> LOG2_WARP_SIZE] = idata2;
//        }
//
//        //wait for warp scans to complete
//        __syncthreads();
//        if( threadIdx.x < (blockDim.x / WARP_SIZE) ){
//            //grab top warp Types
//            Type val = s_Data[threadIdx.x];
//            Type val2 = s_Data2[threadIdx.x];
//            //calculate exclsive scan and write back to shared memory
//            warpScanExclusive2(val,val2, s_Data,s_Data2, size >> LOG2_WARP_SIZE);
//            s_Data[threadIdx.x] = val;
//            s_Data2[threadIdx.x] = val2;
//        }
//
//        //return updated warp scans with exclusive scan results
//        __syncthreads();
//        idata  += s_Data[threadIdx.x >> LOG2_WARP_SIZE];
//        idata2 += s_Data2[threadIdx.x >> LOG2_WARP_SIZE];
//    }
//    else
//        warpScanInclusive2(idata,idata2, s_Data,s_Data2, size);
//
//}
//
//template <typename Type>
//inline __device__ void warpCompareInclusive(Type& idata,Type& idata2,  Type *s_Data, uint size){
//
//     Type* s_Data2;
//    s_Data2 = s_Data + blockDim.x*2;
//    uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
//    s_Data[pos] = 0;
//    s_Data2[pos] = 0;
//    pos += size;
//    s_Data[pos] = idata;
//    s_Data2[pos] = idata2;
//
//    for(uint offset = 1; offset < size; offset <<= 1)
//    {
//        s_Data[pos] =max(s_Data[pos], s_Data[pos - offset]);
//        s_Data2[pos] =min(s_Data2[pos], s_Data2[pos - offset]);
//    }
//
//    idata = s_Data[pos];
//    idata2= s_Data2[pos];
//}
//
//template <typename Type>
//inline __device__ void compareInclusive(Type& idata,Type& idata2,  Type *s_Data, uint size){
//
//     Type* s_Data2;
//    s_Data2 = s_Data + blockDim.x*2;
//    //Bottom-level inclusive warp scan
//    warpCompareInclusive(idata,idata2, s_Data, WARP_SIZE);
//
//    //Save top Types of each warp for exclusive warp scan
//    //sync to wait for warp scans to complete (because s_Data is being overwritten)
//    __syncthreads();
//    if( (threadIdx.x & (WARP_SIZE - 1)) == (WARP_SIZE - 1) )
//    {
//        s_Data[threadIdx.x >> LOG2_WARP_SIZE] = idata;
//        s_Data2[threadIdx.x >> LOG2_WARP_SIZE] = idata2;
//    }
//
//    //wait for warp scans to complete
//    __syncthreads();
//    if( threadIdx.x < (blockDim.x / WARP_SIZE) ){
//        //grab top warp Types
//        Type val = s_Data[threadIdx.x];
//        Type val2 = s_Data2[threadIdx.x];
//        //calculate exclsive scan and write back to shared memory
//        warpCompareInclusive(val,val2, s_Data, size >> LOG2_WARP_SIZE);
//        s_Data[threadIdx.x] =val;
//        s_Data2[threadIdx.x] =val2;
//    }
//
//    //return updated warp scans with exclusive scan results
//    __syncthreads();
//    idata=max(idata,s_Data[threadIdx.x >> LOG2_WARP_SIZE]) ;
//    idata2=min(idata2,s_Data2[threadIdx.x >> LOG2_WARP_SIZE]) ;
//
//}
//
//
//
//
//inline __device__ uint4 scan4Inclusive(uint4 idata4,  uint *s_Data, uint size){
//    //Level-0 exclusive scan
//    idata4.y += idata4.x;
//    idata4.z += idata4.y;
//    idata4.w += idata4.z;
//
//
//    //Level-1 exclusive scan
//    uint oval = scan1Exclusive(idata4.w, s_Data, size / 4);
//
//    idata4.x += oval;
//    idata4.y += oval;
//    idata4.z += oval;
//    idata4.w += oval;
//
//
//    return idata4;
//}
//
////Exclusive vector scan: the array to be scanned is stored
////in local thread memory scope as uint4
//inline __device__ uint4 scan4Exclusive(uint4 idata4,  uint *s_Data, uint size){
//    uint4 odata4 = scan4Inclusive(idata4, s_Data, size);
//    odata4.x -= idata4.x;
//    odata4.y -= idata4.y;
//    odata4.z -= idata4.z;
//    odata4.w -= idata4.w;
//    return odata4;
//}
//
//////////////////////////////////////////////////////////////////////////////////
//// Scan kernels
//////////////////////////////////////////////////////////////////////////////////
//__global__ void scanExclusiveShared(
//        uint4 *d_Dst,
//        uint4 *d_Src,
//        uint size
//){
//    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];
//
//    uint pos = blockIdx.x * blockDim.x + threadIdx.x;
//
//    //Load data
//    uint4 idata4 = d_Src[pos];
//
//    //Calculate exclusive scan
//    uint4 odata4 = scan4Exclusive(idata4, s_Data, size);
//
//    //Write back
//    d_Dst[pos] = odata4;
//}
//
//
//
////Exclusive scan of top elements of bottom-level scans (4 * THREADBLOCK_SIZE)
//__global__ void scanExclusiveShared2(
//        uint *d_Buf,
//        uint *d_Dst,
//        uint *d_Src,
//        uint N,
//        uint arrayLength
//){
//    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];
//
//    //Skip loads and stores for inactive threads of last threadblock (pos >= N)
//    uint pos = blockIdx.x * blockDim.x + threadIdx.x;
//
//    //Load top elements
//    //Convert results of bottom-level scan back to inclusive
//    uint idata = 0;
//    if(pos < N)
//        idata =
//                d_Dst[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos] +
//                d_Src[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos];
//
//    //Compute
//    uint odata = scan1Exclusive(idata, s_Data, arrayLength);
//
//    //Avoid out-of-bound access
//    if(pos < N)
//        d_Buf[pos] = odata;
//}
//
//
////Final step of large-array scan: combine basic inclusive scan with exclusive scan of top elements of input arrays
//__global__ void uniformUpdate(
//        uint4 *d_Data,
//        uint *d_Buffer
//){
//    __shared__ uint buf;
//    uint pos = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if(threadIdx.x == 0)
//        buf = d_Buffer[blockIdx.x];
//    __syncthreads();
//
//    uint4 data4 = d_Data[pos];
//    data4.x += buf;
//    data4.y += buf;
//    data4.z += buf;
//    data4.w += buf;
//    d_Data[pos] = data4;
//}
//
//////////////////////////////////////////////////////////////////////////////////
//// Interface function
//////////////////////////////////////////////////////////////////////////////////
////Derived as 32768 (max power-of-two gridDim.x) * 4 * THREADBLOCK_SIZE
////Due to scanExclusiveShared<<<>>>() 1D block addressing
//const uint MAX_BATCH_ELEMENTS = 64 * 1048576;
//const uint MIN_SHORT_ARRAY_SIZE = 4;
//const uint MAX_SHORT_ARRAY_SIZE = 4 * THREADBLOCK_SIZE;
//const uint MIN_LARGE_ARRAY_SIZE = 8 * THREADBLOCK_SIZE;
//const uint MAX_LARGE_ARRAY_SIZE = 4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE;
//
////Internal exclusive scan buffer
//static uint *d_Buf;
//
//void initScan(void){
//    HANDLE_ERROR( hipMalloc((void **)&d_Buf, (MAX_BATCH_ELEMENTS / (4 * THREADBLOCK_SIZE)) * sizeof(uint)) );
//}
//
//void closeScan(void){
//    HANDLE_ERROR( hipFree(d_Buf) );
//}
//
//static uint factorRadix2(uint& log2L, uint L){
//    if(!L){
//        log2L = 0;
//        return 0;
//    }else{
//        for(log2L = 0; (L & 1) == 0; L >>= 1, log2L++);
//        return L;
//    }
//}
//
//static uint iDivUp(uint dividend, uint divisor){
//    return ( (dividend % divisor) == 0 ) ? (dividend / divisor) : (dividend / divisor + 1);
//}
//
//
//size_t scanExclusiveShort(
//        uint *d_Dst,
//        uint *d_Src,
//        uint batchSize,
//        uint arrayLength
//){
//    //Check power-of-two factorization
//    uint log2L;
//    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
//    assert( factorizationRemainder == 1 );
//
//    //Check supported size range
//    assert( (arrayLength >= MIN_SHORT_ARRAY_SIZE) && (arrayLength <= MAX_SHORT_ARRAY_SIZE) );
//
//    //Check total batch size limit
//    assert( (batchSize * arrayLength) <= MAX_BATCH_ELEMENTS );
//
//    //Check all threadblocks to be fully packed with data
//    assert( (batchSize * arrayLength) % (4 * THREADBLOCK_SIZE) == 0 );
//
//    scanExclusiveShared<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
//            (uint4 *)d_Dst,
//            (uint4 *)d_Src,
//            arrayLength
//    );
////    getLastCudaError("scanExclusiveShared() execution FAILED\n");
//    HANDLE_ERROR(hipPeekAtLastError());
//
//    return THREADBLOCK_SIZE;
//}
//
//size_t scanExclusiveLarge(
//        uint *d_Dst,
//        uint *d_Src,
//        uint batchSize,
//        uint arrayLength
//){
//    //Check power-of-two factorization
//    uint log2L;
//    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
//    assert( factorizationRemainder == 1 );
//
//    //Check supported size range
//    assert( (arrayLength >= MIN_LARGE_ARRAY_SIZE) && (arrayLength <= MAX_LARGE_ARRAY_SIZE) );
//
//    //Check total batch size limit
//    assert( (batchSize * arrayLength) <= MAX_BATCH_ELEMENTS );
//
//    scanExclusiveShared<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
//            (uint4 *)d_Dst,
//            (uint4 *)d_Src,
//            4 * THREADBLOCK_SIZE
//    );
////    getLastCudaError("scanExclusiveShared() execution FAILED\n");
//    HANDLE_ERROR(hipPeekAtLastError());
//
//    //Not all threadblocks need to be packed with input data:
//    //inactive threads of highest threadblock just don't do global reads and writes
//    const uint blockCount2 = iDivUp( (batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE );
//    scanExclusiveShared2<<< blockCount2, THREADBLOCK_SIZE>>>(
//            (uint *)d_Buf,
//            (uint *)d_Dst,
//            (uint *)d_Src,
//            (batchSize * arrayLength) / (4 * THREADBLOCK_SIZE),
//            arrayLength / (4 * THREADBLOCK_SIZE)
//    );
////    getLastCudaError("scanExclusiveShared2() execution FAILED\n");
//    HANDLE_ERROR(hipPeekAtLastError());
//
//    uniformUpdate<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
//            (uint4 *)d_Dst,
//            (uint  *)d_Buf
//    );
////    getLastCudaError("uniformUpdate() execution FAILED\n");
//    HANDLE_ERROR(hipPeekAtLastError());
//
//    return THREADBLOCK_SIZE;
//}
//
/////////////////////////////////////////////////////////////////////////////////////////7
/////// work derived by NVIDIA Corporation
/////////////////////////////////////////////////////////////////////////////////////////
//
///*
// *
// * scan.cu
// *
// * Copyright © 2012-2015 Emanuele Manca
// *
// **********************************************************************************************
// **********************************************************************************************
// *
// 	This file is part of CUDA-Quicksort.
//
//    CUDA-Quicksort is free software: you can redistribute it and/or modify
//    it under the terms of the GNU General Public License as published by
//    the Free Software Foundation, either version 3 of the License, or
//    (at your option) any later version.
//
//    CUDA-Quicksort is distributed in the hope that it will be useful,
//    but WITHOUT ANY WARRANTY; without even the implied warranty of
//    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//    GNU General Public License for more details.
//
//    You should have received a copy of the GNU General Public License
//    along with CUDA-Quicksort.
//
//    If not, see http://www.gnu.org/licenses/gpl-3.0.txt and http://www.gnu.org/copyleft/gpl.html
//
//
//  **********************************************************************************************
//  **********************************************************************************************
// *
// * Contact: Ing. Emanuele Manca
// *
// * Department of Electrical and Electronic Engineering,
// * University of Cagliari,
// * P.zza D’Armi, 09123, Cagliari, Italy
// *
// * email: emanuele.manca@diee.unica.it
// *
// *
// * this software uses the cutil library of NVIDIA CUDA SDK
// *
// * This software contains source code provided by NVIDIA Corporation
// * license: http://developer.download.nvidia.com/licenses/general_license.txt
// *
// * This function are derived by NVIDIA Corporation:
// *
// * 	 1. scanInclusiveLarge()
// * 	 2. scanInclusiveShared()
// * 	 3. scanInclusiveShared2()
// * 	 4. scanInclusiveShort
// *
// *
// *
// */
//
//
//__global__ void scanInclusiveShared2(
//        uint *d_Buf,
//        uint *d_Dst,
//        uint N,
//        uint arrayLength
//){
//    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];
//
//    //Skip loads and stores for inactive threads of last threadblock (pos >= N)
//    uint pos = blockIdx.x * blockDim.x + threadIdx.x;
//
//    //Load top elements
//    //Convert results of bottom-level scan back to inclusive
//    uint idata = 0;
//    if(pos < N)
//        idata =  d_Dst[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos];
//
//    //Compute
//    uint odata = scan1Exclusive(idata, s_Data, arrayLength);
//
//    //Avoid out-of-bound access
//    if(pos < N)
//        d_Buf[pos] = odata;
//}
//
//__global__ void scanInclusiveShared(
//        uint4 *d_Dst,
//        uint4 *d_Src,
//        uint size
//){
//    __shared__ uint s_Data[2 * THREADBLOCK_SIZE];
//
//    uint pos = blockIdx.x * blockDim.x + threadIdx.x;
//
//    //  if(pos<warpSize*(size/warpSize+1))
//    {
//        //Load data
//        uint4 idata4 = d_Src[pos];
//
//        //Calculate exclusive scan
//        uint4 odata4 = scan4Inclusive(idata4, s_Data, size);
//
//        //Write back
//        d_Dst[pos] = odata4;
//    }
//}
//
//size_t scanInclusiveShort(
//        uint *d_Dst,
//        uint *d_Src,
//        uint batchSize,
//        uint arrayLength
//){
//    //Check power-of-two factorization
//    uint log2L;
//    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
//    assert( factorizationRemainder == 1 );
//
//    //Check supported size range
//    //  assert( (arrayLength >= MIN_SHORT_ARRAY_SIZE) && (arrayLength <= MAX_SHORT_ARRAY_SIZE) );
//
//    //Check total batch size limit
//    // assert( (batchSize * arrayLength) <= MAX_BATCH_ELEMENTS );
//
//    //Check all threadblocks to be fully packed with data
//    //assert( (batchSize * arrayLength) % (4 * THREADBLOCK_SIZE) == 0 );
//    int blocks=(batchSize * arrayLength + 4 * THREADBLOCK_SIZE-1 )/ (4 * THREADBLOCK_SIZE);
//    scanInclusiveShared<<<blocks, THREADBLOCK_SIZE>>>(
//            (uint4 *)d_Dst,
//            (uint4 *)d_Src,
//            arrayLength
//    );
////    getLastCudaError("scanExclusiveShared() execution FAILED\n");
//    HANDLE_ERROR(hipPeekAtLastError());
//
//    return THREADBLOCK_SIZE;
//}
//
//size_t scanInclusiveLarge(
//        uint *d_Dst,
//        uint *d_Src,
//        uint batchSize,
//        uint arrayLength
//){
//    //Check power-of-two factorization
//    uint log2L;
//    uint factorizationRemainder = factorRadix2(log2L, arrayLength);
//    assert( factorizationRemainder == 1 );
//
//    //Check supported size range
//    //assert( (arrayLength >= MIN_LARGE_ARRAY_SIZE) && (arrayLength <= MAX_LARGE_ARRAY_SIZE) );
//
//    //Check total batch size limit
//    //assert( (batchSize * arrayLength) <= MAX_BATCH_ELEMENTS );
//
//    scanInclusiveShared<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
//            (uint4 *)d_Dst,
//            (uint4 *)d_Src,
//            4 * THREADBLOCK_SIZE
//    );
////    getLastCudaError("scanExclusiveShared() execution FAILED\n");
//    HANDLE_ERROR(hipPeekAtLastError());
//
//    //Not all threadblocks need to be packed with input data:
//    //inactive threads of highest threadblock just don't do global reads and writes
//    const uint blockCount2 = iDivUp( (batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE );
//    scanInclusiveShared2<<< blockCount2, THREADBLOCK_SIZE>>>(
//            (uint *)d_Buf,
//            (uint *)d_Dst,
//            (batchSize * arrayLength) / (4 * THREADBLOCK_SIZE),
//            arrayLength / (4 * THREADBLOCK_SIZE)
//    );
////    getLastCudaError("scanExclusiveShared2() execution FAILED\n");
//    HANDLE_ERROR(hipPeekAtLastError());
//
//    uniformUpdate<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE>>>(
//            (uint4 *)d_Dst,
//            (uint  *)d_Buf
//    );
////    getLastCudaError("uniformUpdate() execution FAILED\n");
//    HANDLE_ERROR(hipPeekAtLastError());
//
//    return THREADBLOCK_SIZE;
//}
//
//
//template <typename Type>
//__global__ void quick(Type* indata,Type* buffer,  Partition<Type>* partition, Block<Type>* bucket )
//{
//
//    __shared__ Type sh_out[1024];
//
//    __shared__ uint start1,end1;
//    __shared__ uint left,right;
//
//    int tix = threadIdx.x;
//
//    uint start  = partition[blockIdx.x].from;
//    uint end    = partition[blockIdx.x].end;
//    Type pivot  = partition[blockIdx.x].pivot;
//    uint nseq   = partition[blockIdx.x].ibucket;
//
//    uint lo=0;
//    uint hi=0;
//
//    Type lmin = 0xffffffff;
//    Type rmax = 0;
//
//    Type d;
//
//
//    // start read on 1° tile and store the coordinates of the items that must
//    // be moved on the left or on the right of the pivot
//
//    if(tix+start<end)
//    {
//        d = indata[tix+start];
//
//        //count items smaller or bigger than the pivot
//        // if d<pivot then ll++ else ll
//        lo=(d<pivot)*(lo+1)+(d>=pivot)*lo;
//        // if d>pivot then lr++ else lr
//        hi=(d<=pivot)*(hi)+(d>pivot)*(hi+1);
//
//        lmin = d;
//        rmax= d;
//    }
//
//    //read and store the coordinates on next tiles for each block
//    for(uint i=tix+start+blockDim.x;i<end;i+=blockDim.x)
//    {
//        Type d= indata[i];
//
//        //count items smaller or bigger than the pivot
//        lo = ( d <  pivot ) *(lo+1) + ( d >= pivot )*lo;
//        hi = ( d <= pivot ) *(hi)   +  (d >  pivot )*(hi+1);
//
//        //compute max and min of tile items
//        lmin = min(lmin,d);
//        rmax = max(rmax,d);
//
//    }
//
//    //compute max and min of every partition
//
//    compareInclusive(rmax,lmin,(Type*) sh_out, blockDim.x);
//
//    __syncthreads();
//
//    if(tix==blockDim.x-1)
//    {
//        //compute absolute max and min for the bucket
//        atomicMax(&bucket[nseq].maxPiv,rmax);
//        atomicMin(&bucket[nseq].minPiv,lmin);
//    }
//
//    __syncthreads();
//
//
//    /*
//     * calculate the coordinates of its assigned item to each thread,
//     * which are necessary to known in which subsequences the item must be copied
//     *
//     */
//    scan1Inclusive2(lo,hi,(uint*) sh_out, blockDim.x);
//    lo = lo-1;
//    hi = SHARED_LIMIT-hi;
//
//
//    if(tix==blockDim.x-1)
//    {
//        left  = lo+1;
//        right = SHARED_LIMIT-hi;
//
//        start1 = atomicAdd(&bucket[nseq].nextbegin,left);
//        end1   = atomicSub(&bucket[nseq].nextend, right);
//    }
//
//    __syncthreads();
//
//
//    //thread blocks write on the shared memory the items smaller and bigger than the first tile's pivot
//    if(tix+start<end)
//    {
//        //items smaller than pivot
//        if(d<pivot)
//        {sh_out[lo]=d; lo--;}
//
//        //items bigger than pivot
//        if(d>pivot)
//        {sh_out[hi]=d; hi++;}
//
//    }
//
//    //thread blocks write on the shared memory the items smaller and bigger than next tiles' pivot
//    for(uint i=start+tix+blockDim.x;i<end;i+=blockDim.x)
//    {
//
//        Type d=indata[i];
//        //items smaller than the pivot
//        if(d<pivot)
//        {sh_out[lo]=d; lo--;}
//
//        //items bigger than the pivot
//        if(d>pivot)
//        {sh_out[hi]=d; hi++;}
//
//    }
//
//    __syncthreads();
//
//    //items smaller and bigger than the pivot already sorted in the shared memory are coalesced written on the global memory
//    //partial results of each thread block stored on the shared memory are merged together in two subsequences within the global memory
//    //coalesced writing of next tiles on the global memory
//    for(uint i=tix ;i<SHARED_LIMIT;i+=blockDim.x)
//    {
//        if(i<left)
//            buffer[start1+i]=sh_out[i];
//
//        if(i>=SHARED_LIMIT-right)
//            buffer[end1+i-SHARED_LIMIT]=sh_out[i];
//    }
//
//}
//
//
//
////this function assigns the attributes to each partition of each bucket
////a thread block is assigned to a specific partition
//template <typename Type>
//__global__ void partitionAssign(struct Block<Type>* bucket,uint* npartitions,struct Partition<Type>* partition)
//{
//    int tx=threadIdx.x;
//    int bx=blockIdx.x;
//
//    uint beg   = bucket[bx].nextbegin;
//    uint end   = bucket[bx].nextend;
//    Type pivot = bucket[bx].pivot;
//
//    uint from;
//    uint to;
//
//    if(bx>0)
//    {
//        from=npartitions[bx-1];
//        to=npartitions[bx];
//    }
//    else
//    {
//        from=0;
//        to=npartitions[bx];
//    }
//
//
//    uint i=tx+from;
//
//    if(i<to )
//    {
//        uint begin=beg+SHARED_LIMIT*tx;
//        partition[i].from=begin;
//        partition[i].end=begin+SHARED_LIMIT;
//        partition[i].pivot=pivot;
//        partition[i].ibucket=bx;
//
//    }
//
//
//    for(uint i=tx+from+blockDim.x;i<to ;i+=blockDim.x)
//    {
//        uint begin=beg+SHARED_LIMIT*(i-from);
//        partition[i].from=begin;
//        partition[i].end=begin+SHARED_LIMIT;
//        partition[i].pivot=pivot;
//        partition[i].ibucket=bx;
//    }
//    __syncthreads();
//    if(tx==0 && to-from>0) partition[to-1].end=end;
//
//
//}
//
////this function enters the pivot value in the central bucket's items
//template <typename Type>
//__global__ void insertPivot(Type* data,struct Block<Type>* bucket,int nbucket)
//{
//
//    Type pivot      = bucket[blockIdx.x].pivot;
//    uint start      = bucket[blockIdx.x].nextbegin;
//    uint end        = bucket[blockIdx.x].nextend;
//    bool is_altered = bucket[blockIdx.x].done;
//
//    if(is_altered && blockIdx.x<nbucket)
//        for(uint j=start+threadIdx.x; j<end; j+=blockDim.x)
//            data[j]=pivot;
//
//
//}
//
//
////this function assigns the new attributes of each bucket
//template <typename Type>
//__global__ void bucketAssign(Block<Type>* bucket,uint*npartitions,int nbucket,int select)
//{
//
//    uint i=blockIdx.x*blockDim.x+threadIdx.x;
//
//    if(i<nbucket){
//        bool is_altered=bucket[i].done;
//        if(is_altered )
//        {
//            //read on i node
//            uint orgbeg = bucket[i].begin;
//            uint from    = bucket[i].nextbegin;
//            uint orgend = bucket[i].end;
//            uint end    = bucket[i].nextend;
//            Type pivot  = bucket[i].pivot;
//            Type minPiv = bucket[i].minPiv;
//            Type maxPiv = bucket[i].maxPiv;
//
//            //compare each bucket's max and min to the pivot
//            Type lmaxpiv = min(pivot,maxPiv);
//            Type rminpiv = max(pivot,minPiv);
//
//            //write on i+nbucket node
//            bucket[i+nbucket].begin = orgbeg;
//            bucket[i+nbucket].nextbegin   = orgbeg;
//            bucket[i+nbucket].nextend    = from;
//            bucket[i+nbucket].end = from;
//            bucket[i+nbucket].pivot  = (minPiv+lmaxpiv)/2;
//
//            //if(select)
//            //	bucket[i+nbucket].done   = (from-orgbeg)>1024;// && (minPiv!=maxPiv);
//            //else
//            bucket[i+nbucket].done   = (from-orgbeg)>1024 && (minPiv!=maxPiv);
//            bucket[i+nbucket].select=select;
//            bucket[i+nbucket].minPiv = 0xffffffffffffffff;
//            bucket[i+nbucket].maxPiv = 0;
//            //bucket[i+nbucket].finish=false;
//
//            //calculate the number of partitions (npartitions) necessary to the i+nbucket bucket
//            if(!bucket[i+nbucket].done)
//                npartitions[i+nbucket] = 0;
//            else npartitions[i+nbucket] = (from-orgbeg+SHARED_LIMIT-1)/SHARED_LIMIT;
//
//            //write on i node
//            bucket[i].begin = end;
//            bucket[i].nextbegin   = end;
//            bucket[i].nextend    = orgend;
//            bucket[i].pivot  = (rminpiv+maxPiv)/2+1;
//
//            //if(select)
//            //bucket[i].done   = (orgend-end)>1024;// && (minPiv!=maxPiv);
//            //	else
//            bucket[i].done   = (orgend-end)>1024 && (minPiv!=maxPiv);
//            bucket[i].select=select;
//            bucket[i].minPiv = 0xffffffffffffffff;
//            bucket[i].maxPiv = 0;
//            //bucket[i].finish=false;
//
//            //calculate the number of partitions (npartitions) necessary to the i-bucket bucket
//            if(!bucket[i].done)
//                npartitions[i]=0;
//            else
//                npartitions[i]=(orgend-end+SHARED_LIMIT-1)/SHARED_LIMIT;
//
//        }
//    }
//
//
//}
//
//
//
//template <typename Type>
//__global__ void init(Type* data,Block<Type>* bucket,uint* npartitions,int size, int nblocks)
//{
//    uint i=blockIdx.x*blockDim.x+threadIdx.x;
//
//    if(i<nblocks)
//    {
//        bucket[i].nextbegin   = 0;
//        bucket[i].begin = 0;
//
//        bucket[i].nextend    = 0 + size*(i==0);
//        bucket[i].end = 0 + size*(i==0);
//        npartitions[i]   = 0;
//        bucket[i].done   = false + i==0;
//        bucket[i].select   = false;
//        bucket[i].maxPiv = 0x0;
//        bucket[i].minPiv = 0xffffffffffffffff;
//        //bucket[i].pivot  = 0+ (i==0)*((min(min(data[0],data[size/2]),data[size-1]) + max(max(data[0],data[size/2]),data[size-1]))/2);
//        bucket[i].pivot = data[size/2];
//    }
//
//}
//
//
//
//template <typename Type>
//void sort(Type *inputData, Type *outputData, uint size, uint threadCount, int device)
//{
//
//    hipSetDevice(device);
//
//    hipGetLastError();
//    //hipDeviceReset();
//
//    hipDeviceProp_t deviceProp;
//    hipGetDeviceProperties(&deviceProp, device);
//
//    Type* ddata;
//    Type* dbuffer;
//
//    Block<Type>* dbucket;
//    struct Partition<Type>* partition;
//    uint* npartitions1,*npartitions2;
//
//    uint*cudaBlocks=(uint*)malloc(4);
//
//    uint blocks = (size + SHARED_LIMIT-1)/SHARED_LIMIT;
//    uint nblock=10*blocks;
//    int partition_max= 262144;
//
//    unsigned long long int total = partition_max*sizeof(Block<Type>) + nblock*sizeof(Partition<Type>) + 2*partition_max*sizeof(uint) +3*(size)*sizeof(Type);
//
//    printf("\nINFO: Device Memory consumed is %.3f GB out of %.3f GB of available memory\n", ((double)total/GIGA), (double)deviceProp.totalGlobalMem/GIGA);
//
//    //Allocating and initializing CUDA arrays
////	sdkCreateTimer(&htimer);
//    HANDLE_ERROR( hipMalloc  ((void**)&dbucket   , partition_max*sizeof(Block<Type>)) );
//    HANDLE_ERROR( hipMalloc  ((void**)&partition , nblock*sizeof(Partition<Type>)) ); //nblock
//
//    HANDLE_ERROR(hipMalloc((void**)&npartitions1,partition_max*sizeof(uint)) );
//    HANDLE_ERROR(hipMalloc((void**)&npartitions2,partition_max*sizeof(uint)) );
//
//    HANDLE_ERROR(hipMalloc((void**)&dbuffer,(size)*sizeof(Type)));
//    HANDLE_ERROR(hipMalloc((void**)&ddata  ,(size)*sizeof(Type)));
//
//    HANDLE_ERROR(hipMemcpy(ddata, inputData, size*sizeof(Type), hipMemcpyHostToDevice) );
//
//    initScan();
//
//    //setting GPU Cache
//    hipFuncSetCacheConfig(reinterpret_cast<const void*>(init<Type>),           	 hipFuncCachePreferL1);
//    hipFuncSetCacheConfig(reinterpret_cast<const void*>(insertPivot<Type>),    	 hipFuncCachePreferL1);
//    hipFuncSetCacheConfig(reinterpret_cast<const void*>(bucketAssign<Type>),   	 hipFuncCachePreferL1);
//    hipFuncSetCacheConfig(reinterpret_cast<const void*>(partitionAssign<Type>), 	 hipFuncCachePreferL1);
//    hipFuncSetCacheConfig(reinterpret_cast<const void*>(quick<Type>),          	 hipFuncCachePreferShared);
//    hipFuncSetCacheConfig(reinterpret_cast<const void*>(globalBitonicSort<Type>),  hipFuncCachePreferShared);
//
//
//    HANDLE_ERROR(hipDeviceSynchronize());
////    sdkResetTimer(&htimer);
////    sdkStartTimer(&htimer);
//
//    //initializing bucket array: initial attributes for each bucket
//    init<Type><<<(nblock+255)/256,256>>>(ddata,dbucket,npartitions1,size,partition_max);
//
//
//    uint nbucket     = 1;
//    uint numIterations  = 0;
//    bool inputSelect = true;
//
//    *cudaBlocks=blocks;
//    HANDLE_ERROR(hipDeviceSynchronize());
////	getLastCudaError("init() execution FAILED\n");
//    HANDLE_ERROR( hipMemcpy(&npartitions2[0], cudaBlocks,  sizeof(uint), hipMemcpyHostToDevice) );
//
//
//    // beginning of the first phase
//    // this phase goes on until the size of the buckets is comparable to the SHARED_LIMIT size
//    while(1)
//    {
//
//        /*
//         *       	---------------------    Pre-processing: Partitioning    ---------------------
//         *
//         * buckets are further divided in partitions based on their size
//         * the number of partitions needed for each subsequence is determined by the number of elements which can be
//         * processed by each thread block.
//         *
//         * the number of partitions (npartitions) for each block will depend on the shared memory size (SHARED_LIMIT)
//         *
//         */
//
//        if(numIterations>0)
//        {	//1024 is the shared memory limit of scanInclusiveShort()
//            if(nbucket<=1024)
//                scanInclusiveShort(npartitions2, npartitions1, 1, nbucket);
//            else
//                scanInclusiveLarge(npartitions2, npartitions1, 1, nbucket);
//
//            HANDLE_ERROR( hipMemcpy(cudaBlocks, &npartitions2[nbucket-1],  sizeof(uint), hipMemcpyDeviceToHost) );
//        }
//
//        if(*cudaBlocks==0)
//            break;
//
//
//        /*
//         *  ---------------------     step 1    ---------------------
//         *
//         * 	A thread block is assigned to each different partition
//         * 	each partition is assigned coordinates, pivot and ....
//         */
//
//
//        partitionAssign<Type><<<nbucket,1024>>>(dbucket,npartitions2,partition);
//        hipDeviceSynchronize();
////		getLastCudaError("partitionAssign() execution FAILED\n");
//
//        /*
//              ---------------------    step 2a    ---------------------
//
//              in this function each thread block creates two subsequences
//              to divide the items in the partition whose value is lower than
//             the pivot value, from the items whose value is higher than the pivot value
//         */
//
//        if(inputSelect)
//            quick<Type><<<*cudaBlocks,threadCount>>>(ddata,dbuffer,partition,dbucket);
//        else
//            quick<Type><<<*cudaBlocks,threadCount>>>(dbuffer,ddata,partition,dbucket);
//        hipDeviceSynchronize();
////		getLastCudaError("quick() execution FAILED\n");
//
//        //step 2b: this function enters the pivot value in the central bucket's items
//        insertPivot<Type><<<nbucket,512>>>(ddata,dbucket,nbucket);
//
//
//        //step 3: parameters are assigned, linked to the two new buckets created in step 2
//        bucketAssign<Type><<<(nbucket+255)/256,256>>>(dbucket,npartitions1,nbucket,inputSelect);
//        hipDeviceSynchronize();
////		getLastCudaError("insertPivot() or bucketAssign() execution FAILED\n");
//
//        nbucket*=2;
//
//        inputSelect = !inputSelect;
//        numIterations++;
//        if(nbucket>deviceProp.maxGridSize[0])
//            break;
//        //if(numIterations==18) break;
//    }
//
//    /*
//     * start second phase:
//     * now the size of the buckets is such that they can be entirely processed by a thread block
//     *
//     */
//
//    if(nbucket>deviceProp.maxGridSize[0])
//        fprintf(stderr, "ERROR: CUDA-Quicksort can't terminate sorting as the block threads needed to finish it are more than the Maximum x-dimension of FERMI GPU thread blocks. Please use Kepler GPUs as the Maximum x-dimension of their thread blocks is much higher\n");
//    else
//        globalBitonicSort<Type><<<nbucket,512,0>>>(ddata,dbuffer,dbucket,inputSelect);
//
//    HANDLE_ERROR(hipDeviceSynchronize());
////	getLastCudaError("globalBitonicSort() execution FAILED\n");
//
////	sdkStopTimer(&htimer);
////    *wallClock=sdkGetTimerValue(&htimer);
//
//
//    // Copy the final result to the CPU in the outputData array
//    HANDLE_ERROR(hipMemcpy(outputData, ddata, size*sizeof(Type), hipMemcpyDeviceToHost) );
//
//    // release resources
//    HANDLE_ERROR( hipFree(ddata) );
//    HANDLE_ERROR( hipFree(dbuffer) );
//    HANDLE_ERROR( hipFree(dbucket) );
//    HANDLE_ERROR( hipFree(npartitions2));
//    HANDLE_ERROR( hipFree(npartitions1));
//    free(cudaBlocks);
//
//    closeScan();
//    return ;
//}
//
//
//
//void CUDA_Quicksort(Metrics* inputData, Metrics* outputData, uint dataSize, uint threadCount /*, int Device, double* wallClock*/)
//{
//
//    hipDeviceProp_t deviceProp;
//    hipGetDeviceProperties(&deviceProp, 0);
//
//    if(deviceProp.major<2)
//    {
//        fprintf(stderr, "Error: the GPU device %d has a Compute Capability of %d.%d, while a Compute Capability of 2.x is required to run the code\n",
//                0, deviceProp.major, deviceProp.minor);
//
//        int deviceCount;
//        hipGetDeviceCount(&deviceCount);
//
//        fprintf(stderr, "       the Host system has the following GPU devices:\n");
//
//        for (int device = 0; device < deviceCount; device++) {
//
//            fprintf(stderr, "\t  the GPU device %d is a %s, with Compute Capability %d.%d\n",
//                    device, deviceProp.name, deviceProp.major, deviceProp.minor);
//        }
//
//        return;
//    }
//
//    sort<Metrics>(inputData,outputData, dataSize,threadCount,0);
//}
////
////void CUDA_Quicksort_64(double* inputData,double* outputData, uint dataSize, uint threadCount, int Device, double* wallClock)
////{
////
////    hipDeviceProp_t deviceProp;
////    hipGetDeviceProperties(&deviceProp, Device);
////
////    if(deviceProp.major<2)
////    {
////        fprintf(stderr, "Error: the GPU device %d has a Compute Capability of %d.%d, while a Compute Capability of 2.x is required to run the code\n",
////                Device, deviceProp.major, deviceProp.minor);
////
////        int deviceCount;
////        hipGetDeviceCount(&deviceCount);
////
////        fprintf(stderr, "       the Host system has the following GPU devices:\n");
////
////        for (int device = 0; device < deviceCount; device++) {
////
////            fprintf(stderr, "\t  the GPU device %d is a %s, with Compute Capability %d.%d\n",
////                    device, deviceProp.name, deviceProp.major, deviceProp.minor);
////        }
////
////        return;
////    }
////
////    sort<double>(inputData,outputData, dataSize,threadCount,Device);
////
////}