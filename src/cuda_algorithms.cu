#include "hip/hip_runtime.h"
//
// Created by breucking on 28.12.21.
//

#include <stdlib.h>
#include <time.h>
#include <c++/9/chrono>
#include <string.h>
#include <stdio.h>
#include <string>
#include <uuid/uuid.h>
#include "../src/cuda_algorithms.cuh"
#include "cuda_algorithms.cuh"

#include <cstdlib>

#include <hip/hip_runtime.h>

#include <iostream>

#include <math.h>
#include <chrono>
#include <ctime>

#include "graphcode.h"
#include "cudahelper.cuh"
#include "helper.h"

#include "reduce.cuh"

#include <hip/hip_runtime_api.h>


/**
 * Calc Metrices is a simple example to compare two NxN matrices
 * @param data pinter to vectorized matrix
 * @param comparedata pointer to vectorized matrix
 * @param matrixSize dimension of the NxN matrix
 * @param numOfNonZeroEdges pointer to array to store the values for the non zero edges comparison
 * @param edgeMetricCount pointer to array to store the values for the edge metric comparison
 * @param edgeType pointer to array to store the values for the edge type metric comparison
 */


void checkNxNConstraint(const GraphCode &gc1, const GraphCode &gc2);

void demoCalculateGCsOnCuda(int NUMBER_OF_GCS, unsigned int dictCounter, const unsigned short *gcMatrixData,
                            const unsigned int *gcDictData, const unsigned int *gcMatrixOffsets,
                            const unsigned int *gcDictOffsets, const unsigned int *gcMatrixSizes);

__global__ void
calcMetrices(unsigned short int *data, unsigned short int *comparedata, unsigned long noItems,
             unsigned int *numOfNonZeroEdges, unsigned int *edgeMetricCount, unsigned int *edgeType) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int /*offset*/ tid = x + y * blockDim.x * gridDim.x;

    numOfNonZeroEdges[tid] = 0;
    edgeMetricCount[tid] = 0;
    edgeType[tid] = 0;

    if (tid > noItems) {
        return;
    }

    if (x != y && data[tid] != 0) {
        numOfNonZeroEdges[tid] = 1;
        if (comparedata[tid] != 0) {
            edgeMetricCount[tid] = 1;
            if (data[tid] == comparedata[tid]) {
                edgeType[tid] = 1;
            }

        }
    }

}


void print_d_array(unsigned int *d_array, int len) {
    int *h_array = new int[len];
    HANDLE_ERROR(hipMemcpy(h_array, d_array, sizeof(int) * len, hipMemcpyDeviceToHost));
    for (int i = 0; i < len; ++i) {
        std::cout << h_array[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_array;
}

Metrics demoCudaLinearMatrixMemoryCudaReduceSum(GraphCode json1, GraphCode json2) {
    checkNxNConstraint(json1, json2);
    hipProfilerStart();
    int items1 = pow(json1.dict->size(), 2);

    // Prep for cuda


    unsigned short int *gpu_inputMatrix1;
    unsigned short int *gpu_inputMatrix2;
    unsigned int *darr_edge_metric_count;
    unsigned int *darr_num_of_non_zero_edges;
    unsigned int *darr_edge_type;

    auto start = std::chrono::system_clock::now();

    HANDLE_ERROR(hipMalloc((void **) &gpu_inputMatrix1, sizeof(unsigned short int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &gpu_inputMatrix2, sizeof(unsigned short int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &darr_num_of_non_zero_edges, sizeof(unsigned int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &darr_edge_metric_count, sizeof(unsigned int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &darr_edge_type, sizeof(unsigned int) * items1));

    // Transfer data from host to device memory
    HANDLE_ERROR(
            hipMemcpy(gpu_inputMatrix1, json1.matrix, sizeof(unsigned short int) * items1, hipMemcpyHostToDevice));
    HANDLE_ERROR(
            hipMemcpy(gpu_inputMatrix2, json2.matrix, sizeof(unsigned short int) * items1, hipMemcpyHostToDevice));

    dim3 block;
    dim3 grid;

    int width = json1.dict->size();
    calcKernelLaunchConfig(width, block, grid);

    //HANDLE_ERROR(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, calcMetrices, 0, 0));

    // calculation
    auto loaded = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = loaded - start;

    if (G_DEBUG)
        std::cout << "elapsed time: " << elapsed_seconds.count()
                  << std::endl;

    //int q = sqrt((float) items1);
    calcMetrices<<<grid, block>>>(gpu_inputMatrix1, gpu_inputMatrix2, items1,
                                  darr_num_of_non_zero_edges,
                                  darr_edge_metric_count,
                                  darr_edge_type
    );



    //printf("CUDA error %s\n",hipGetErrorString(hipPeekAtLastError()));
    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    auto end = std::chrono::system_clock::now();
    elapsed_seconds = end - start;
    std::time_t end_time = std::chrono::system_clock::to_time_t(end);

    if (G_DEBUG) {
        std::cout << "finished computation at " << std::ctime(&end_time)
                  << "elapsed time: " << elapsed_seconds.count() << "s\n";
        elapsed_seconds = end - loaded;
        std::cout << "Computation time: " << elapsed_seconds.count() << "s\n";
    }
    auto mem_start = std::chrono::system_clock::now();

    unsigned int gts_edge_metric_count = gpu_sum_reduce(darr_edge_metric_count, items1);
    unsigned int gts_edge_type = gpu_sum_reduce(darr_edge_type, items1);
    unsigned int gts_num_of_non_zero_edges = gpu_sum_reduce(darr_num_of_non_zero_edges, items1);

    if (G_DEBUG) {
        std::cout << gts_num_of_non_zero_edges << std::endl;
    }

    auto mem_end = std::chrono::system_clock::now();
    if (G_DEBUG) {
        elapsed_seconds = mem_end - mem_start;
        std::cout << "Sum Reduce time: " << elapsed_seconds.count() << "s\n";
    }


    HANDLE_ERROR(hipFree(gpu_inputMatrix1));
    HANDLE_ERROR(hipFree(gpu_inputMatrix2));
    HANDLE_ERROR(hipFree(darr_edge_metric_count));
    HANDLE_ERROR(hipFree(darr_num_of_non_zero_edges));
    HANDLE_ERROR(hipFree(darr_edge_type));

    // Result reduction
    int num_of_non_zero_edges = gts_num_of_non_zero_edges;
    int edge_metric_count = gts_edge_metric_count;
    int edgeTypeCount = gts_edge_type;

    std::string gc1Dict[json1.dict->size()];

    int sim = 0;
    int n = 0;
    for (const auto &item: *json1.dict) {
        std::string str = item;
        gc1Dict[n++] = str;


        for (const auto &item2: *json2.dict) {
            if (str == item2) {
                //std::cout << "Match" << std::endl;
                sim++;
            }
        }
    }

    // Calculate metrices
    float node_metric = (float) sim / (float) json1.dict->size();


    float edge_metric = 0.0;
    if (num_of_non_zero_edges > 0)
        edge_metric = (float) edge_metric_count / (float) num_of_non_zero_edges;

    float edge_type_metric = 0.0;
    if (edge_metric_count > 0)
        edge_type_metric = (float) edgeTypeCount / (float) edge_metric_count;

    if (G_DEBUG)
        std::cout << "Similarity: " << " value: " << node_metric << std::endl;
    if (G_DEBUG)
        std::cout << "Recommendation: " << " value: " << edge_metric << std::endl;


    if (G_DEBUG) {
        auto metrics_end = std::chrono::system_clock::now();
        elapsed_seconds = metrics_end - mem_end;
        std::cout << "Metrics Management time: " << elapsed_seconds.count() << "s\n";
    }

    Metrics m;
    m.similarity = node_metric;
    m.recommendation = edge_metric;
    m.inferencing = edge_type_metric;

    hipProfilerStop();
    return m;

}

void checkNxNConstraint(const GraphCode &gc1, const GraphCode &gc2) {

    if (gc1.dict->size() != gc2.dict->size()) {
        std::cout << "Graph Codes need to have same size" << std::endl;
        exit(71);
    }
    bool result = std::equal(gc1.dict->begin(), gc1.dict->end(), gc2.dict->begin());

    if (!result) {
        std::cout << "Graph Codes need to have same dict elements" << std::endl;
        exit(71);
    }
}


Metrics demoCudaLinearMatrixMemory(GraphCode json1, GraphCode json2) {
    checkNxNConstraint(json1, json2);

    int items1 = pow(json1.dict->size(), 2);

    // Prep for cuda


    unsigned short int *gpu_inputMatrix1;
    unsigned short int *gpu_inputMatrix2;
    unsigned int *darr_edge_metric_count;
    unsigned int *darr_num_of_non_zero_edges;
    unsigned int *darr_edge_type;

    auto start = std::chrono::system_clock::now();

    HANDLE_ERROR(hipMalloc((void **) &gpu_inputMatrix1, sizeof(unsigned short int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &gpu_inputMatrix2, sizeof(unsigned short int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &darr_num_of_non_zero_edges, sizeof(unsigned int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &darr_edge_metric_count, sizeof(unsigned int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &darr_edge_type, sizeof(unsigned int) * items1));

    // Transfer data from host to device memory
    HANDLE_ERROR(
            hipMemcpy(gpu_inputMatrix1, json1.matrix, sizeof(unsigned short int) * items1, hipMemcpyHostToDevice));
    HANDLE_ERROR(
            hipMemcpy(gpu_inputMatrix2, json2.matrix, sizeof(unsigned short int) * items1, hipMemcpyHostToDevice));

    dim3 block;
    dim3 grid;

    calcKernelLaunchConfig(json1.dict->size(), block, grid);

    // calculation
    auto loaded = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = loaded - start;

    if (G_DEBUG)
        std::cout << "elapsed time: " << elapsed_seconds.count()
                  << std::endl;


    calcMetrices<<<grid, block>>>(gpu_inputMatrix1, gpu_inputMatrix2, items1,
                                  darr_num_of_non_zero_edges,
                                  darr_edge_metric_count,
                                  darr_edge_type
    );

    HANDLE_ERROR(hipPeekAtLastError());

    auto end = std::chrono::system_clock::now();
    elapsed_seconds = end - start;
    std::time_t end_time = std::chrono::system_clock::to_time_t(end);

    if (G_DEBUG) {
        std::cout << "finished computation at " << std::ctime(&end_time)
                  << "elapsed time: " << elapsed_seconds.count() << "s\n";
        elapsed_seconds = end - loaded;
        std::cout << "Computation time: " << elapsed_seconds.count() << "s\n";
    }


    int *arrEdgeTypeMetricCount;
    HANDLE_ERROR(hipHostMalloc((void **) &arrEdgeTypeMetricCount, sizeof(int) * items1));

    int *arr_edge_metric_count;
    HANDLE_ERROR(hipHostMalloc((void **) &arr_edge_metric_count, sizeof(int) * items1));

    int *arr_num_of_non_zero_edges;
    HANDLE_ERROR(hipHostMalloc((void **) &arr_num_of_non_zero_edges, sizeof(int) * items1));


    HANDLE_ERROR(hipMemcpy(arr_num_of_non_zero_edges, darr_num_of_non_zero_edges, sizeof(int) * items1,
                            hipMemcpyDeviceToHost));

    HANDLE_ERROR(
            hipMemcpy(arr_edge_metric_count, darr_edge_metric_count, sizeof(int) * items1, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipMemcpy(arrEdgeTypeMetricCount, darr_edge_type, sizeof(int) * items1, hipMemcpyDeviceToHost));


    auto mem_end = std::chrono::system_clock::now();
    if (G_DEBUG) {
        elapsed_seconds = mem_end - end;
        std::cout << "Memory Management time: " << elapsed_seconds.count() << "s\n";
    }


    HANDLE_ERROR(hipFree(gpu_inputMatrix1));
    HANDLE_ERROR(hipFree(gpu_inputMatrix2));
    HANDLE_ERROR(hipFree(darr_edge_metric_count));
    HANDLE_ERROR(hipFree(darr_num_of_non_zero_edges));
    HANDLE_ERROR(hipFree(darr_edge_type));

    // Result reduction
    int num_of_non_zero_edges = 0;
    int edge_metric_count = 0;
    int edgeTypeCount = 0;
    for (int i = 0; i < items1; i++) {
        if (arr_edge_metric_count[i] == 1) {
            edge_metric_count++;
        }
        if (arr_num_of_non_zero_edges[i] == 1) {
            num_of_non_zero_edges++;
        }
        if (arrEdgeTypeMetricCount[i] == 1) {
            edgeTypeCount++;
        }
    }

    std::string gc1Dict[json1.dict->size()];

    int sim = 0;
    int n = 0;
    for (const auto &item: *json1.dict) {
        //std::cout << item.value() << "\n";
        std::string str = item;
        gc1Dict[n++] = str;


        for (const auto &item2: *json2.dict) {
            if (str == item2) {
                //std::cout << "Match" << std::endl;
                sim++;
            }
        }
    }

    // Calculate metrices
    float node_metric = (float) sim / (float) json1.dict->size();


    float edge_metric = 0.0;
    if (num_of_non_zero_edges > 0)
        edge_metric = (float) edge_metric_count / (float) num_of_non_zero_edges;

    float edge_type_metric = 0.0;
    if (edge_metric_count > 0)
        edge_type_metric = (float) edgeTypeCount / (float) edge_metric_count;

    if (G_DEBUG)
        std::cout << "Similarity: " << " value: " << node_metric << std::endl;
    if (G_DEBUG)
        std::cout << "Recommendation: " << " value: " << edge_metric << std::endl;


    if (G_DEBUG) {
        auto metrics_end = std::chrono::system_clock::now();
        elapsed_seconds = metrics_end - mem_end;
        std::cout << "Metrics Management time: " << elapsed_seconds.count() << "s\n";
    }

    Metrics m;
    m.similarity = node_metric;
    m.recommendation = edge_metric;
    m.inferencing = edge_type_metric;


    hipHostFree(arrEdgeTypeMetricCount);
    hipHostFree(arr_num_of_non_zero_edges);
    hipHostFree(arr_edge_metric_count);

    return m;

}


Metrics demoCudaLinearMatrixMemory(json json1, json json2) {
    //checkNxNConstraint(json1, json2);

    json gc1Dictionary;
    int numberOfElements1;
    long items1;
    unsigned short int *inputMatrix1;

    convertGc2Cuda(json1, gc1Dictionary, numberOfElements1, items1, inputMatrix1);


    json gc2Dictionary;
    int numberOfElements2;
    long items2;
    unsigned short int *inputMatrix2;
    convertGc2Cuda(json2, gc2Dictionary, numberOfElements2, items2, inputMatrix2);

    // Prep for cuda


    unsigned short int *gpu_inputMatrix1;
    unsigned short int *gpu_inputMatrix2;
    unsigned int *darr_edge_metric_count;
    unsigned int *darr_num_of_non_zero_edges;
    unsigned int *darr_edge_type;

    auto start = std::chrono::system_clock::now();

    HANDLE_ERROR(hipMalloc((void **) &gpu_inputMatrix1, sizeof(unsigned short int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &gpu_inputMatrix2, sizeof(unsigned short int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &darr_num_of_non_zero_edges, sizeof(unsigned int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &darr_edge_metric_count, sizeof(unsigned int) * items1));
    HANDLE_ERROR(hipMalloc((void **) &darr_edge_type, sizeof(unsigned int) * items1));

    // Transfer data from host to device memory
    HANDLE_ERROR(
            hipMemcpy(gpu_inputMatrix1, inputMatrix1, sizeof(unsigned short int) * items1, hipMemcpyHostToDevice));
    HANDLE_ERROR(
            hipMemcpy(gpu_inputMatrix2, inputMatrix2, sizeof(unsigned short int) * items1, hipMemcpyHostToDevice));


    dim3 block;
    dim3 grid;

    int width = numberOfElements1;

    calcKernelLaunchConfig(width, block, grid);

    // calculation
    auto loaded = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = loaded - start;

    if (G_DEBUG)
        std::cout << "elapsed time: " << elapsed_seconds.count()
                  << std::endl;

    calcMetrices<<<grid, block>>>(gpu_inputMatrix1, gpu_inputMatrix2, items1,
                                  darr_num_of_non_zero_edges,
                                  darr_edge_metric_count,
                                  darr_edge_type
    );


    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
    auto end = std::chrono::system_clock::now();

    elapsed_seconds = end - start;
    std::time_t end_time = std::chrono::system_clock::to_time_t(end);

    if (G_DEBUG)
        std::cout << "finished computation at " << std::ctime(&end_time)
                  << "elapsed time: " << elapsed_seconds.count() << "s\n";
    elapsed_seconds = end - loaded;
    if (G_DEBUG)
        std::cout << "Computation time: " << elapsed_seconds.count() << "s\n";

    // Retrieve results
    int *arrEdgeTypeMetricCount = (int *) malloc(sizeof(int) * items1);
    int *arr_edge_metric_count = (int *) malloc(sizeof(int) * items1);
    int *arr_num_of_non_zero_edges = (int *) malloc(sizeof(int) * items1);

    HANDLE_ERROR(hipMemcpy(arr_num_of_non_zero_edges, darr_num_of_non_zero_edges, sizeof(int) * items1,
                            hipMemcpyDeviceToHost));
    HANDLE_ERROR(
            hipMemcpy(arr_edge_metric_count, darr_edge_metric_count, sizeof(int) * items1, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipMemcpy(arrEdgeTypeMetricCount, darr_edge_type, sizeof(int) * items1, hipMemcpyDeviceToHost));

    HANDLE_ERROR(hipFree(gpu_inputMatrix1));
    HANDLE_ERROR(hipFree(gpu_inputMatrix2));
    HANDLE_ERROR(hipFree(darr_edge_metric_count));
    HANDLE_ERROR(hipFree(darr_num_of_non_zero_edges));
    HANDLE_ERROR(hipFree(darr_edge_type));


    free(inputMatrix1);
    free(inputMatrix2);

    // Result reduction
    int num_of_non_zero_edges = 0;
    int edge_metric_count = 0;
    int edgeTypeCount = 0;
    for (int i = 0; i < items1; i++) {
        if (arr_edge_metric_count[i] == 1) {
            edge_metric_count++;
        }
        if (arr_num_of_non_zero_edges[i] == 1) {
            num_of_non_zero_edges++;
        }
        if (arrEdgeTypeMetricCount[i] == 1) {
            edgeTypeCount++;
        }
    }

    std::string gc1Dict[gc1Dictionary.size()];

    int sim = 0;
    int n = 0;
    for (const auto &item: gc1Dictionary.items()) {
        std::string str = item.value().get<std::string>();
        gc1Dict[n++] = str;


        for (const auto &item2: gc2Dictionary.items()) {
            if (str == item2.value()) {
                sim++;
            }
        }
    }

    // Calculate metrices
    float node_metric = (float) sim / (float) gc1Dictionary.size();


    float edge_metric = 0.0;
    if (num_of_non_zero_edges > 0)
        edge_metric = (float) edge_metric_count / (float) num_of_non_zero_edges;

    float edge_type_metric = 0.0;
    if (edge_metric_count > 0)
        edge_type_metric = (float) edgeTypeCount / (float) edge_metric_count;

    if (G_DEBUG)
        std::cout << "Similarity: " << " value: " << node_metric << std::endl;
    if (G_DEBUG)
        std::cout << "Recommendation: " << " value: " << edge_metric << std::endl;

    Metrics m;
    m.similarity = node_metric;
    m.recommendation = edge_metric;
    m.inferencing = edge_type_metric;

    free(arrEdgeTypeMetricCount);
    free(arr_num_of_non_zero_edges);
    free(arr_edge_metric_count);

    return m;

}

void calcKernelLaunchConfig(int width, dim3 &block, dim3 &grid) {

    if (width > 32) {
        int gridSize = ceil(width / 32.0);

        block = dim3(32, 32, 1);
        grid = dim3(gridSize, gridSize, 1);

    } else {

        block = dim3(width, width);
        grid = (1);
    }
}

void convertGc2Cuda(const json &gcq, json &gc1Dictionary, int &numberOfElements, long &items,
                    unsigned short int *&inputMatrix) {
    gc1Dictionary = gcq["dictionary"];
    numberOfElements = gc1Dictionary.size();
    items = numberOfElements * numberOfElements;// Transform to data structures for calculations
    int *matrix1;
    matrix1 = (int *) malloc(sizeof(int) * numberOfElements * numberOfElements);

    convertDict2Matrix(numberOfElements, matrix1, gcq["matrix"]);

    inputMatrix = (unsigned short int *) malloc(sizeof(unsigned short int) * numberOfElements * numberOfElements);

    int count = 0;
    for (int i = 0; i < numberOfElements; i++)
        for (int j = 0; j < numberOfElements; j++) {
            inputMatrix[count++] = matrix1[i * numberOfElements + j]; //matrix1[i][j];
        }
    free(matrix1);
}

Metrics demoCalculateSimilaritySequentialOrdered(GraphCode gc1, GraphCode gc2) {

    int sim = 0;

    unsigned short *matrix1 = gc1.matrix;
    unsigned short *matrix2 = gc2.matrix;

    for (const auto &item: *gc1.dict) {
        for (const auto &item2: *gc2.dict) {
            if (item == item2) {
                sim++;
            }
        }
    }
    int num_of_non_zero_edges = 0;
    int edge_metric_count = 0;
    int edge_type = 0;

    for (int i = 0; i < gc1.dict->size(); i++) {
        for (int j = 0; j < gc1.dict->size(); j++) {

            if (i != j && matrix1[i * gc1.dict->size() + j] != 0) {
                num_of_non_zero_edges++;

                int position1 = i;
                int position2 = j;
                if (position1 == -1 || position2 == -1) {
                    continue;
                }

                int edge = matrix2[position1 * gc1.dict->size() + position2];//matrix2[position1][position2];
                if (edge != 0) {
                    edge_metric_count++;
                }
                if (edge == matrix1[i * gc1.dict->size() + j]) {
                    edge_type++;
                }

            }
        }
    }

    float node_metric = (float) sim / (float) gc1.dict->size();
    float edge_metric = 0.0;
    if (num_of_non_zero_edges > 0)
        edge_metric = (float) edge_metric_count / (float) num_of_non_zero_edges;
    float edge_type_metric = 0.0;
    if (edge_metric_count > 0)
        edge_type_metric = (float) edge_type / (float) edge_metric_count;

    Metrics metrics;
    metrics.similarity = node_metric;
    metrics.recommendation = edge_metric;
    metrics.inferencing = edge_type_metric;
    return metrics;

}


Metrics demoCalculateSimilaritySequentialOrdered(json gc1, json gc2) {
    int sim = 0;

    json gc1Dictionary;
    int numberOfElements1;
    long items1;
    unsigned short int *matrix1;

    convertGc2Cuda(gc1, gc1Dictionary, numberOfElements1, items1, matrix1);

    json gc2Dictionary;
    int numberOfElements2;
    long items2;
    unsigned short int *matrix2;
    convertGc2Cuda(gc2, gc2Dictionary, numberOfElements2, items2, matrix2);

    std::vector<std::string> dict2;
    for (const auto &item2: gc2Dictionary.items()) {
        dict2.push_back(item2.value().get<std::string>());
    }


    for (const auto &item: gc1Dictionary.items()) {

        std::string str = item.value().get<std::string>();

        for (const auto &item2: gc2Dictionary.items()) {
            if (str == item2.value()) {
                sim++;
            }
        }

    }
    int num_of_non_zero_edges = 0;
    int edge_metric_count = 0;
    int edge_type = 0;

    for (int i = 0; i < gc1Dictionary.size(); i++) {
        for (int j = 0; j < gc1Dictionary.size(); j++) {

            if (i != j && matrix1[i * gc1Dictionary.size() + j] != 0) {
                num_of_non_zero_edges++;

                int position1 = i;
                int position2 = j;
                if (position1 == -1 || position2 == -1) {
                    continue;
                }

                int edge = matrix2[position1 * gc1Dictionary.size() + position2];//matrix2[position1][position2];
                if (edge != 0) {
                    edge_metric_count++;
                }
                if (edge == matrix1[i * gc1Dictionary.size() + j]) {
                    edge_type++;
                }

            }
        }
    }

    float node_metric = (float) sim / (float) gc1Dictionary.size();
    float edge_metric = 0.0;
    if (num_of_non_zero_edges > 0)
        edge_metric = (float) edge_metric_count / (float) num_of_non_zero_edges;
    float edge_type_metric = 0.0;
    if (edge_metric_count > 0)
        edge_type_metric = (float) edge_type / (float) edge_metric_count;

    Metrics metrics;
    metrics.similarity = node_metric;
    metrics.recommendation = edge_metric;
    metrics.inferencing = edge_type_metric;
    return metrics;

}


__global__ void compare2(unsigned short *gcMatrixData, unsigned int *gcDictData, unsigned int *gcMatrixOffsets,
                         unsigned int *gcMatrixSizes, unsigned int *gcDictOffsets, int gcToCompare,
                         Metrics *metrics) {
    int index = blockIdx.x;
    int gc1 = gcToCompare;
    int gc2 = index;

    int sim = 0;
    int elements = sqrtf((float) gcMatrixSizes[gc1]);

    for (int i = 0; i < elements; i++) {
        for (int j = 0; j < elements; j++) {
            unsigned int off1 = gcDictOffsets[gc1];
            unsigned int off2 = gcDictOffsets[gc2];
            unsigned int d1 = gcDictData[off1 + i];
            unsigned int d2 = gcDictData[off2 + j];
            if (d1 == d2) {
                sim++;
            }
        }
    }

    int num_of_non_zero_edges = 0;
    int edge_metric_count = 0;
    int edge_type = 0;


    for (int i = 0; i < elements; i++) {
        for (int j = 0; j < elements; j++) {

            if (i != j && gcMatrixData[gcMatrixOffsets[gc1] + i * elements + j] != 0) {
                num_of_non_zero_edges++;

                int position1 = i;
                int position2 = j;
                if (position1 == -1 || position2 == -1) {
                    continue;
                }
                int edge = gcMatrixData[gcMatrixOffsets[gc2] + position1 * elements +
                                        position2];
                if (edge != 0) {
                    edge_metric_count++;
                }
                if (edge == gcMatrixData[gcMatrixOffsets[gc1] + i * elements + j]) {
                    edge_type++;
                }

            }
        }
    }
    metrics[index].similarity = 0.0;
    metrics[index].recommendation = 0.0;
    metrics[index].inferencing = 0.0;
    metrics[index].similarity = (float) sim / (float) elements;
    if (num_of_non_zero_edges > 0) {
        /*edge_metric*/ metrics[index].recommendation = (float) edge_metric_count / (float) num_of_non_zero_edges;
    }
    if (edge_metric_count > 0) {
        /*edge_type_metric*/ metrics[index].inferencing = (float) edge_type / (float) edge_metric_count;
    }
}

void demoCalculateGCsOnCuda(int NUMBER_OF_GCS, unsigned int dictCounter, const unsigned short *gcMatrixData,
                            const unsigned int *gcDictData, const unsigned int *gcMatrixOffsets,
                            const unsigned int *gcDictOffsets, const unsigned int *gcMatrixSizes, int gcQueryPosition) {
    //------------
    // CUDA prep
    //------------


    unsigned short *d_gcMatrixData;
    unsigned int *d_gcDictData;
    unsigned int *d_gcMatrixOffsets;
    unsigned int *d_gcMatrixSizes;
    unsigned int *d_gcDictOffsets;
    Metrics *d_result;

    long md_size = 0;
    for (int i = 0; i < NUMBER_OF_GCS; i++) {
        md_size += gcMatrixSizes[i];
    }// ;



    HANDLE_ERROR(hipMalloc((void **) &d_gcMatrixData, md_size * sizeof(unsigned short)));
    HANDLE_ERROR(hipMalloc((void **) &d_gcDictData, dictCounter * sizeof(unsigned int)));
    HANDLE_ERROR(hipMalloc((void **) &d_gcMatrixOffsets, NUMBER_OF_GCS * sizeof(unsigned int)));
    HANDLE_ERROR(hipMalloc((void **) &d_gcMatrixSizes, NUMBER_OF_GCS * sizeof(unsigned int)));
    HANDLE_ERROR(hipMalloc((void **) &d_gcDictOffsets, NUMBER_OF_GCS * sizeof(unsigned int)));
    HANDLE_ERROR(hipMalloc((void **) &d_result, NUMBER_OF_GCS * sizeof(Metrics)));


    HANDLE_ERROR(
            hipMemcpy(d_gcMatrixData, gcMatrixData, md_size * sizeof(unsigned short), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_gcDictData, gcDictData, dictCounter * sizeof(unsigned int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_gcMatrixOffsets, gcMatrixOffsets, NUMBER_OF_GCS * sizeof(unsigned int),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(
            hipMemcpy(d_gcMatrixSizes, gcMatrixSizes, NUMBER_OF_GCS * sizeof(unsigned int), hipMemcpyHostToDevice));
    HANDLE_ERROR(
            hipMemcpy(d_gcDictOffsets, gcDictOffsets, NUMBER_OF_GCS * sizeof(unsigned int), hipMemcpyHostToDevice));

    auto start = std::chrono::system_clock::now();

    compare2<<<NUMBER_OF_GCS, 1>>>(d_gcMatrixData,
                                   d_gcDictData,
                                   d_gcMatrixOffsets,
                                   d_gcMatrixSizes,
                                   d_gcDictOffsets,
                                   gcQueryPosition,
                                   d_result);

    HANDLE_ERROR(hipPeekAtLastError());
    HANDLE_ERROR(hipDeviceSynchronize());
    auto end = std::chrono::system_clock::now();

    std::chrono::duration<double> elapsed_seconds = end - start;
    time_t end_time = std::chrono::system_clock::to_time_t(end);

    std::cout << "finished CUDA computation at " << ctime(&end_time)
              << "elapsed time: " << elapsed_seconds.count() << "s\n";


    Metrics *result = (Metrics *) malloc(NUMBER_OF_GCS * sizeof(Metrics));
    HANDLE_ERROR(hipMemcpy(result, d_result, NUMBER_OF_GCS * sizeof(Metrics), hipMemcpyDeviceToHost));

    for (int i = 0; i < NUMBER_OF_GCS; i++) {


//        std::cout << "Result (" << i << ") "
//                  << "Similarity " << result[i].similarity << "; "
//                  << "Recommendation " << result[i].recommendation << "; "
//                  << "Inference " << result[i].inferencing << "; "
//                  << std::endl;

//        assert(result[i].similarity == 1);
//        assert(result[i].recommendation == 0.5);
//        assert(result[i].inferencing == 0);
    }
    HANDLE_ERROR(hipFree(d_gcMatrixData));
    HANDLE_ERROR(hipFree(d_gcDictData));
    HANDLE_ERROR(hipFree(d_gcMatrixOffsets));
    HANDLE_ERROR(hipFree(d_gcMatrixSizes));
    HANDLE_ERROR(hipFree(d_result));
}