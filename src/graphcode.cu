#include "hip/hip_runtime.h"
//
// Created by Patrick Steinert on 16.10.21.

#include "graphcode.h"


#include <thread>
#include <chrono>
#include <ctime>

#include <iostream>
#include <fstream>

#include <nlohmann/json.hpp>


#ifdef __HIPCC__
#include <experimental/filesystem>
namespace fs = std::experimental::filesystem;
#elif __GNUC__
  #include <features.h>
  #if __GNUC_PREREQ(8,0)
  //      If  gcc_version >= 8.0
    #include <filesystem>
    namespace fs = std::filesystem;

  #  else
  //       Else gcc_version < 8.0
    #include <experimental/filesystem>
    namespace fs = std::experimental::filesystem;
  #endif
#else
//    If not gcc
  #include <filesystem>
  namespace fs = std::__fs::filesystem;
#endif

//0. Ein File haben wir (gcQuery), dazu die similarity berehcnen
//
//1. Für jedes Element in der Collection:
//1.a Einlesen der Files in einem Directory (for each file)
//1.b Für jedes File similarity berechnen
//   2. Für jedes Element im Dictionary (Annotation) der gcQuery
//   2. a-> für jedes element in der Matrix
//        wenn der wert in QCquery != 0
//            num_of_non_zero_edges = 1
//        -> check anderes GC element:
//            wenn es die Begriffe im Dict nicht gibt skip
//            else
//                -> wenn matrix wert != 0 dann edge_metric_count = 1
//                -> wenn matrix wert == gcQuery Wert (beziehungstyp identisch) edge_type =
//  2. b -> Kalkulation der
//1.c Sortieren


using json = nlohmann::json;


int getPosition(std::string string, std::vector<std::string> dictionary);

void convertDict2Matrix(int size, int *destMatrix, json jsonMatrix);
#define N 1000

__global__ void vector_add(int *a, int *b, int *c) {
    int tid = blockIdx.x;
    if(tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}




void myThreadFun(int i, const std::vector<std::string> &files, std::vector<json> *arr)
{


    for (int j = 0; j < files.size(); j++) {

        //std::cout << entry.path() << std::endl;
        try {
            //std::ifstream ifs(entry.path());
            std::ifstream ifs(files.at(j));
            json jf = json::parse(ifs);

            //std::cout << jf["dictionary"] << std::endl;
            arr->push_back(jf);

        } catch (json::exception &e) {
            std::cerr << e.what() << '\n';
        }
        if (j % 10000 == 0)
            std::cout << "Thread " << i << ": Status " << j << std::endl;
    }


}

void gmaf::GraphCode::loadGraphCodes(char *directory, int limit,  std::vector<json> *arr) {



    std::vector<std::string> files;


    for (const auto &entry: fs::directory_iterator(directory)) {
        files.push_back(entry.path().string());
    }

    int s = 4;
    int x;
    if (limit > files.size()) {
        x = files.size() / s;
    } else {
        x = limit / s;
    }
    std::vector<std::thread> threads;

    std::vector<json> tmp_jsons[s];

    for (int i = 0; i < s; i++) {
        std::vector<std::string> sub(&files[i * x + 1], &files[(i + 1) * x]);

        threads.push_back(std::thread(myThreadFun, i, sub, &tmp_jsons[i]));
    }

    for (auto &th: threads) {
        th.join();
    }

    arr->reserve(tmp_jsons[0].size() + tmp_jsons[1].size() + tmp_jsons[2].size() + tmp_jsons[3].size());
    arr->insert(arr->end(), tmp_jsons[0].begin(), tmp_jsons[0].end());
    arr->insert(arr->end(), tmp_jsons[1].begin(), tmp_jsons[1].end());
    arr->insert(arr->end(), tmp_jsons[2].begin(), tmp_jsons[2].end());
    arr->insert(arr->end(), tmp_jsons[3].begin(), tmp_jsons[3].end());



//        if (i++ > limit) {
//            break;
//        }


}


int calculateSimilaritySequential(json gc1, json gc2, float *results) {
    json gc1Dictionary = gc1["dictionary"];
    json gc2Dictionary = gc2["dictionary"];

    std::string gc1Dict[gc1Dictionary.size()];

    int n = 0;

    int sim = 0;



    int matrix1[gc1Dictionary.size()][gc1Dictionary.size()];
    convertDict2Matrix(gc1Dictionary.size(), (int *) matrix1, gc1["matrix"]);

    int matrix2[gc2Dictionary.size()][gc2Dictionary.size()];
    convertDict2Matrix(gc2Dictionary.size(), (int *) matrix2, gc2["matrix"]);


    std::vector<std::string> dict2;
    for (const auto &item2: gc2Dictionary.items()) {
        dict2.push_back(item2.value().get<std::string>());
    }


    for (const auto &item: gc1Dictionary.items()) {
        //std::cout << item.value() << "\n";
        std::string str = item.value().get<std::string>();
        gc1Dict[n++] = str;


        for (const auto &item2: gc2Dictionary.items()) {
            if (str == item2.value()) {
                //std::cout << "Match" << std::endl;
                sim++;
            }
        }
    }
    int num_of_non_zero_edges = 0;
    int edge_metric_count = 0;
    int edge_type = 0;
    for (int i = 0; i < gc1Dictionary.size(); i++) {
        for (int j = 0; j < gc1Dictionary.size(); j++) {

            if (i != j && matrix1[i][j] != 0) {
                num_of_non_zero_edges++;

                int position1 = getPosition(gc1Dict[i], dict2);
                int position2 = getPosition(gc1Dict[j], dict2);
                //std::cout << "Pos " << position1 << " " << position2 << std::endl;
                if (position1 == -1 || position2 == -1) {
                    continue;
                }

                int edge = matrix2[position1][position2];
                if (edge != 0) {
                    edge_metric_count++;
                }
                if (edge == matrix1[i][j]) {
                    edge_type++;
                }

            }
        }
    }

    float node_metric = (float) sim / (float) gc1Dictionary.size();
    float edge_metric = 0.0;
    if (num_of_non_zero_edges > 0)
        edge_metric = (float) edge_metric_count / (float) num_of_non_zero_edges;
    float edge_type_metric = 0.0;
    if (edge_metric_count > 0)
        edge_type_metric = (float) edge_type / (float) edge_metric_count;

    results[0] = node_metric;
    results[1] = edge_metric;
    results[2] = edge_type_metric;
    return 0;

}

void convertDict2Matrix(int size, int *destMatrix, json jsonMatrix) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {

           //destMatrix[i][j] = jsonMatrix.at(i).at(j);
            *((destMatrix+i*size) + j) = jsonMatrix.at(i).at(j);
        }
    }
}

int calculateSimilarityCuda(json gc1, json gc2, float *results) {
    json gc1Dictionary = gc1["dictionary"];
    json gc2Dictionary = gc2["dictionary"];

    std::string gc1Dict[gc1Dictionary.size()];

    int n = 0;

    int sim = 0;




    int matrix1[gc1Dictionary.size()][gc1Dictionary.size()];
    convertDict2Matrix(gc1Dictionary.size(), (int *) matrix1, gc1["matrix"]);

    int matrix2[gc2Dictionary.size()][gc2Dictionary.size()];
    convertDict2Matrix(gc2Dictionary.size(), (int *) matrix2, gc2["matrix"]);



    std::vector<std::string> dict2;
    for (const auto &item2: gc2Dictionary.items()) {
        dict2.push_back(item2.value().get<std::string>());
    }


    for (const auto &item: gc1Dictionary.items()) {
        //std::cout << item.value() << "\n";
        std::string str = item.value().get<std::string>();
        gc1Dict[n++] = str;


        for (const auto &item2: gc2Dictionary.items()) {
            if (str == item2.value()) {
                //std::cout << "Match" << std::endl;
                sim++;
            }
        }
    }
    int num_of_non_zero_edges = 0;
    int edge_metric_count = 0;
    int edge_type = 0;
    for (int i = 0; i < gc1Dictionary.size(); i++) {
        for (int j = 0; j < gc1Dictionary.size(); j++) {
            int a[N], b[N], c[N];
            int *d_a, *d_b, *d_c;
            //float *d_a;

            //a = (float*)malloc(sizeof(float) * N);

            // Allocate device memory for a
            hipMalloc((void**)&d_a, sizeof(int) * N);
            hipMalloc((void**)&d_b, sizeof(int) * N);
            hipMalloc((void**)&d_c, sizeof(int) * N);

            for(int z=0; z<N; z++) {
                a[z] = -z;
                b[z] = z * z;
            }

            // Transfer data from host to device memory
            hipMemcpy(d_a, a, sizeof(int) * N, hipMemcpyHostToDevice);
            hipMemcpy(d_b, b, sizeof(int) * N, hipMemcpyHostToDevice);
            hipMemcpy(d_c, c, sizeof(int) * N, hipMemcpyHostToDevice);


            vector_add<<<N,1>>>(d_a, d_b, d_c);

            hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);



            // Cleanup after kernel execution
            hipFree(d_a);
            hipFree(d_b);
            hipFree(d_c);

            if (i != j && matrix1[i][j] != 0) {
                num_of_non_zero_edges++;

                int position1 = getPosition(gc1Dict[i], dict2);
                int position2 = getPosition(gc1Dict[j], dict2);
                //std::cout << "Pos " << position1 << " " << position2 << std::endl;
                if (position1 == -1 || position2 == -1) {
                    continue;
                }

                int edge = matrix2[position1][position2];
                if (edge != 0) {
                    edge_metric_count++;
                }
                if (edge == matrix1[i][j]) {
                    edge_type++;
                }

            }
        }
    }

    float node_metric = (float) sim / (float) gc1Dictionary.size();
    float edge_metric = 0.0;
    if (num_of_non_zero_edges > 0)
        edge_metric = (float) edge_metric_count / (float) num_of_non_zero_edges;
    float edge_type_metric = 0.0;
    if (edge_metric_count > 0)
        edge_type_metric = (float) edge_type / (float) edge_metric_count;

    results[0] = node_metric;
    results[1] = edge_metric;
    results[2] = edge_type_metric;
    return 0;

}


int getPosition(std::string string, std::vector<std::string> dictionary) {
    for (int i = 0; i < dictionary.size(); i++) {
        if (dictionary.at(i) == string) {
            return i;
        }
    }
    return -1;
}

void gmaf::GraphCode::foo() {}

void gmaf::GraphCode::calculateSimilarityV(int index, json *gcQuery, std::vector<json> *compares, int start, int end) {
    for (int i = start; i < end; i++) {

        std::cout << "Idx " << index << " i " << i << " limit(" << end << ")" << std::endl;

        float resultMetrics[3];
        calculateSimilaritySequential(*gcQuery, compares->at(i), resultMetrics);

        std::cout << "Similarity " << resultMetrics[0] << std::endl;
        std::cout << "Recommendation " << resultMetrics[1] << std::endl;
        std::cout << "Inferencing " << resultMetrics[2] << std::endl;
    }
}

