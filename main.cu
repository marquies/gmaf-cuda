#include "hip/hip_runtime.h"
#include <iostream>
#include <unistd.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "main.cpp"

__global__ void kernel( void ) {}

/*
int main() {

             kernel<<<1,1>>>();
    std::vector<json> arr;

    thrust::host_vector<int> h_vec(32 << 20);
    loadGraphCodes((char *) "../graphcodes/", &arr);


    std::cout << "loaded " << arr.size() << " graph code files." << std::endl;

//icudaMalloc( (void**)&dev_c, sizeof(int) )

   // while(true) {
        for (int i = 1; i < arr.size(); i++) {

            float resultMetrics[3];
            calculateSimilarity(arr.at(0), arr.at(i), resultMetrics);



            std::cout << "Similarity " << resultMetrics[0] << std::endl;
            std::cout << "Recommendation " << resultMetrics[1] << std::endl;
            std::cout << "Inferencing " << resultMetrics[2] << std::endl;
        }
   // }

    return 0;
}
*/
